#include "hip/hip_runtime.h"
#include "header/fillMatrix.h"

/*
 * Calcul la representation binaire d'un int avec une taille fixée
 * Parametres :
 *     tab : Le tableau contenant la representation binaire
 *     n : le nombre a decomposer
 *     size : la taille de la representation binaire
 */
__device__ __host__ void intToBinWithSize(int *tab, int n, int size) {
    int i;
    for(i = 0; i < size; i++)
        tab[i] = (n >> ((size - 1) - i)) & 0x1;
}

/*__device__ int *intToBin(int n, int *tab_size) {
	int size = ((int) ((double) floor((double) log(n)/ (double) log(2)))) + 1;
	int *tab = (int *) malloc(size * sizeof(int));
	intToBinWithSize(tab, n, size);
	*tab_size = size;
	return tab;
}*/

/*
 * Parametres :
 *     yList : La liste des y de l'ensemble des couples
 *     premList : La liste des premiers construite au départ
 *     size : La taille de la representation binaire du thread le plus grand
 *     result : le vecteur trouvé, NULL s'il est pas trouvé
 * Contraintes :
 *     A lancer avec taille(yList) blocks et 2^(taille(premList)) threads
 * Resultat :
 *     La matrice des vecteurs qui sont décomposition des y de yList en facteurs premiers stockée dans result
 */
__global__ void fillMatrix(int *yList, int *premList, int size, int **result) {
	__shared__ volatile int found;
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	if (threadId == 0) {
		found = 0;
		//result[blockId] = (int *) malloc(size * sizeof(int));
	}
	__syncthreads();
	int *listCoeff = (int *) malloc(size * sizeof(int));
	intToBinWithSize(listCoeff, threadId, size);
	int res = 1;
	for (int i = 0; i < size; i++) {
		res *= (int) ((double) pow((double) premList[i], (double) listCoeff[i]));
	}
	if (yList[blockId] == res) {
		found = 1;
		result[blockId] = listCoeff;
	}
	__syncthreads();
	if (found == 0) {
		result[blockId] = NULL;
	}
}


