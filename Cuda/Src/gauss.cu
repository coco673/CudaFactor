#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


void print_matrix(int **matrix, int size) {
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%d\t", matrix[i][j]);
		}
		printf("\n");
	}
}

/*int lcm(int x,int y) {
	int t = 0;
	while (y != 0) {
		t = y;
		y = x % y;
		x = t;
	}
	return x;
}


void gauss_elimination(int **matrix, int size) {
	int l = 0;
	int d1 = 0;
	int d2 = 0;
	for (int i = 0; i < size-1; i++) {
		for (int j = i+1; j < size; j++) {
			l = lcm(matrix[i][i], matrix[j][i]);
			if (l != 0 && (matrix[i][i] != 0 && matrix[j][i] != 0)) {
				l = (matrix[i][i] * matrix[j][i]) / l;
				d1 = l / matrix[i][i];
				d2 = l / matrix[j][i];
				matrix[j][i] = 0;
				for (int k = i+1; k < size; k++) {
					matrix[j][k] = (d2 * matrix[j][k]) - (d1 * matrix[i][k]);
				}
			}
		}
	}
}

void gauss_jordan_elimination(int **gauss_matrix, int size) {
	int l = 0;
	int d1 = 0;
	int d2 = 0;
	for (int i = size-1; i > 0; i--) {
		for (int j = i-1; j >= 0; j--) {
			l = lcm(gauss_matrix[i][i], gauss_matrix[j][i]);
			if (l != 0 && (gauss_matrix[i][i] != 0 && gauss_matrix[j][i] != 0)) {
				l = (gauss_matrix[i][i] * gauss_matrix[j][i]) / l;
				d1 = l / gauss_matrix[i][i];
				d2 = l / gauss_matrix[j][i];
				for (int k = 0; k <= size; k++) {
					gauss_matrix[j][k] = (d2 * gauss_matrix[j][k]) - (d1 * gauss_matrix[i][k]);
				}
			}
		}
	}
}

int *kernel(int **matrix, int size) {
	int *kernel = (int *) malloc(size * sizeof(int));
	gauss_elimination(matrix, size);
	gauss_jordan_elimination(matrix, size);
	for (int i = 0; i < size; i++) {
		kernel[i] = matrix[i][i];
	}
	return kernel;
}*/

int *gaussjordan_noyau(int **matrix, int size) {
	int pivo,j,temp,a,k,l;
	int nl = size;
	int nc = size;
	//on met des 0 sous la diagonale
	int jc = 0;
	int jl = 0;
	// on traite toutes les colonnes
	while (jc < nc && jl < nl) {
		//choix du pivot que l'on veut mettre en M[jl,jc]
		k = jl;
		while (matrix[k][jc] == 0 && k < nl-1) {
			k = k+1;
		}
		//on ne fait la suite que si on a pivo!=0
		if (matrix[k][jc] != 0) {
			pivo = matrix[k][jc];
			//echange de la ligne jl et de la ligne k
			for (l = jc; l < nc; l++){
				temp = matrix[jl][l];
				matrix[jl][l] = matrix[k][l];
				matrix[k][l] = temp;
			}
			//fin du choix du pivot qui est M[jl,jc]
			//on met 1 sur la diagonale de la colonne jc
			for (l = 0; l < nc; l++) {
				matrix[jl][l] = matrix[jl][l] / pivo;
			}
			//on met des 0 au dessus de la diagonale
			// de la colonne jc
			for (k = 0; k < jl; k++) {
				a = matrix[k][jc];
				for (l = 0; l < nc; l++) {
					matrix[k][l] = matrix[k][l] - matrix[jl][l] * a;
				}
			}
			//on met des 0 sous la diag de la colonne jc
			for (k = jl+1; k < nl; k++) {
				a = matrix[k][jc];
				for (l = jc; l < nc; l++) {
					matrix[k][l] = matrix[k][l] - matrix[jl][l] * a;
				}
			}
		} else {
			//on ajoute une ligne de 0 si ce n'est pas le dernier 0
			if (jl < nc-1) {
				for (j = nl; j > jl; j--) {
					//matrix[j] = matrix[j-1];
					for (int tmp = 0; tmp < size; tmp++) {
						matrix[j][tmp] = matrix[j-1][tmp];
					}
				}
				//M[jl] = makelist(0,1,nc);
				for (int tmp = 0; tmp < size; tmp++) {
					matrix[jl][tmp] = 0;
				}
				nl = nl + 1;
			}
		}
		//ds tous les cas,le numero de colonne et
		//le numero de ligne augmente de 1
		jc = jc + 1;
		jl = jl + 1;
		//il faut faire toutes les colonnes
		if (jl == nl && jl < nc) {
			//matrix[nl] = makelist(0,1,nc);
			for (int tmp = 0; tmp < size; tmp++) {
				matrix[nl][tmp] = 0;
			}
			nl = nl + 1;
		}
	}
	int *noyau = (int *) malloc(size * sizeof(int));
	//on enleve les lignes en trop pour avoir
	//une matrice carree de dim nc
	//on retranche la matrice identite
	//matrix = matrix[0..nc-1] - idn(nc);
	for (int tmp = 0; tmp < nc; tmp++) {
		matrix[tmp][tmp] = matrix[tmp][tmp] - 1;
	}
	for(int j = 0; j < nc; j++){
		if (matrix[j][j] == -1) {
			//noyau = append(noyau,M[0..nc-1,j]);
			for (int tmp = 0; tmp < nc; tmp++) {
				noyau[tmp] = matrix[tmp][j];
			}
		}
	}
	return noyau;
}

int main(int argc, char **argv) {
	int **matrix = (int **) malloc(4 * sizeof(int));
	for (int i = 0; i < 4; i++) {
		matrix[i] = (int *) malloc(4 * sizeof(int));
		for (int j = 0; j < 4; j++) {
			matrix[i][j] = 1;
		}
	}
	matrix[0][1] = 0;
	matrix[1][2] = 0;
	matrix[2][0] = 0;
	matrix[2][2] = 0;
	int *k = gaussjordan_noyau(matrix, 4);
	for (int i = 0; i < 4; i++) {
		printf("%d\n", k[i]);
	}
	return EXIT_SUCCESS;
}

