#include "hip/hip_runtime.h"
/*
 * fillEns.cu
 *
 *  Created on: 3 avr. 2014
 *      Author: groupeDev
 */
#include <stdio.h>
#include "fillEns.h"
#include "prime.h"
#define N 1000

/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ int isInf(uint64_t *list, int size, uint64_t y){
	int i = threadIdx.x;
	int res= 0;
	/*if(i == 0){
		res = 0;
	}*/
	__syncthreads();
	if(i < size){
		for(int j = 0; j <= size;j ++){
			if(list[j] > y){
				res=1;
				break;
			}
		}
		__syncthreads();
	}
	return res;
}
/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__device__ void isBSmoothG(uint64_t *list,int size, uint64_t y,int *result){
	int i =threadIdx.x+blockIdx.x;
	if(i < size){
		int y1 = y;
		if(y1 == 0){
			*result = 0;
		}else{

			for(int j = 0; j< size;j++){
				while(y1 % list[j] == 0){
					y1=y1/list[j];
				}
			}
			__syncthreads();
			if(y1 == 1){
				*result= 1;
			} else {
				*result= 0;
			}
		}
	}
}
/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__device__ void isInEnsembleG(uint64_t *ens, uint64_t y,int size, int *res){
	int i = threadIdx.x+blockIdx.x;
	int found = 0;

	__syncthreads();
	if(i < size){

		for(int j=0;j<size;j++){
			if(getValGPU(ens,j) == y){
				found = 1;
			}
		}
	}
	__syncthreads();
	*res = found;

}

__device__ void setup_kernel ( hiprandState_t *state )
{
	int id = threadIdx.x + blockIdx.x;
	hiprand_init ( clock()+id, id, 0, &state[id] );
}

__device__ void generate( hiprandState_t *globalState, uint64_t *rand, uint64_t nbr, uint64_t racN)
{

	int id = threadIdx.x + blockIdx.x ;
	//int id = threadIdx.x;
	float x;

	hiprandState_t localState = globalState[id];
	for(int n = 0; n < N; n++) {
		x = fmodf(hiprand(&localState),(nbr-racN)) + racN;
	}
	globalState[id] = localState;
	rand[id] = (int) x;
}

__global__ void Generation(hiprandState_t *state,uint64_t nbr, uint64_t sqrtNBR,uint64_t *rand){
	setup_kernel(state);
}


__global__ void fillEnsR(hiprandState_t *state,Couple *R,int *size,uint64_t *Div,int sizeDiv,uint64_t *premList,int k,uint64_t *rand,uint64_t nbr,int *matrix){
	//int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tid=threadIdx.x+blockIdx.x;

	__shared__ int sizeR;
	int bsmooth= -1;
	int present= -1;
	uint64_t x = 0;
	uint64_t y =  0;
	int nbt = 0;
	if(tid == 0){
		sizeR = 0;
	}
	__syncthreads();
	uint64_t sqrtNBR = (uint64_t) sqrtf(nbr);
	do{
		generate(state,rand,nbr,sqrtNBR);

		x = rand[tid];
		y = (x*x) % nbr;
		if(premList == NULL ){
			printf("PrimeList est NULL\n");
		}
		if(k <= 0 ){
			printf("valeur de K <= 0 \n");
		}

		isBSmoothG(premList,k,y,&bsmooth);

		isInEnsembleG(Div,y,sizeDiv,&present);
		++nbt;

		__syncthreads();

	}while(!bsmooth || present);

	__syncthreads();
	R[tid].x = x;
	R[tid].y = y;

	atomicAdd(&sizeR,1);

	int y1 = y;
	for(int j = 0;j<k;j++){
		while(y1%premList[j] == 0){
			y1 = y1 / premList[j];
			matrix[tid*k+j]=(matrix[tid*k+j]+1);
		}
	}

	__syncthreads();
	size[0] = sizeR;
}
