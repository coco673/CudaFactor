#include "hip/hip_runtime.h"
/*
 * fillEns.cu
 *
 *  Created on: 3 avr. 2014
 *      Author: groupeDev
 */
#include <stdio.h>
#include "fillEnsemble.h"
#include "prime.h"
#define N 1000
/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ int isInf(int *list, int size, int y){
	int i = threadIdx.x;
	int res= 0;
	/*if(i == 0){
		res = 0;
	}*/
	__syncthreads();
	if(i < size){
		printf("size = %i\n",size);
		for(int j = 0; j <= size;j ++){
			printf("thread = %i y = %i :: list[%i] = %i j = %i\n",i,y,j,list[j]);
			if(list[j] > y){
				res=1;
				break;
			}
		}
		printf("y = %i::res = %i\n",y,res);
		__syncthreads();
	}
	return res;
}
/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__device__ void isBSmoothG(int *list,int size, int y,int *result){
	int i =threadIdx.x+blockIdx.x;
	if(i < size){
		int y1 = y;
		if(y1 == 0){
			*result = 0;
		}else{

			for(int j = 0; j< size;j++){
				while(y1 % list[j] == 0){
					y1=y1/list[j];
				}
			}
			__syncthreads();
			if(y1 == 1){
				*result= 1;
			} else {
				*result= 0;
			}
		}
	}

}
/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__device__ void isInEnsembleG(ensemble ens, int y,int size, int *res){
	int i = threadIdx.x+blockIdx.x;
	int found = 0;

	printf("debut\n");
	printf("size = %i\n",size);
	__syncthreads();
	if(i < size){

		for(int j=0;j<size;j++){
			if(ens == NULL || ens[j].ind.val == y){
				found = 1;
			}
		}
	}
	__syncthreads();
	*res = found;
	printf("fin ! %i\n",found);
	printf("fin 2! %i\n",*res);

}

__device__ void setup_kernel ( hiprandState_t *state )
{
	int id = threadIdx.x + blockIdx.x;
	//int id = threadIdx.x;
	printf("l h eure de %i\n",clock());
	hiprand_init ( clock()+id, id, 0, &state[id] );
}

__device__ void generate( hiprandState_t *globalState, int *rand, int nbr, int racN)
{

	int id = threadIdx.x + blockIdx.x ;
	//int id = threadIdx.x;
	float x;

	hiprandState_t localState = globalState[id];
	for(int n = 0; n < N; n++) {
		x = fmodf(hiprand(&localState),(nbr-racN)) + racN;
	}
	globalState[id] = localState;
	rand[id] = (int) x;
}

__global__ void Generation(hiprandState_t *state,int nbr, int sqrtNBR,int *rand){
	int i = threadIdx.x + blockIdx.x;
	setup_kernel(state);
}


__global__ void fillEnsR(hiprandState_t *state,ensemble R,int *size,ensemble Div,int *sizeDiv,int *premList,int k,int *rand,int nbr,int *matrix){
	//int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tid=threadIdx.x+blockIdx.x;
	__shared__ int sizeR;
	int bsmooth= -1;
	int present= -1;
	int x = -1;
	int y =  -1;
	int nbt = 0;
	printf("tu print batard !!!!!!\n");
	if(tid == 0){
		sizeR = 0;
	}
	__syncthreads();
	printf("sizeDiv %i\n",*sizeDiv);
	//memset(matrix,0,(k*k)*sizeof(int));
	int sqrtNBR = (int) sqrtf(nbr);
	do{

		generate(state,rand,nbr,sqrtNBR);
		printf("yahou %i::%i\n",rand[tid],tid);

		x = rand[tid];
		y = (x*x) % nbr;

		isBSmoothG(premList,k,y,&bsmooth);
		isInEnsembleG(Div,y,*sizeDiv,&present);
		++nbt;
		__syncthreads();
		printf("je suis le thread %i et je suis au %i tours de boucle ::::: bsmooth %i pres = %i\n",tid,nbt,bsmooth,present);
	}while(!bsmooth || present);
	__syncthreads();
	printf("je suis le gentil  BON THREAD numéro : %i , j'ai fait  %i tours \n", tid, nbt);

		printf("ok\n");
	printf("la size de R est %i\n",sizeof(R));
	R[tid].ind.couple.x = x;
	printf("pourquoi ?? size = %i\n",sizeR);
	R[tid].ind.couple.y = y;
	atomicAdd(&sizeR,1);
	printf("x = %i y = %i size = %i\n",R[tid].ind.couple.x,R[tid].ind.couple.y,sizeR);
	int y1 = y;
	for(int j = 0;j<k;j++){
		while(y1%premList[j] == 0){
			y1 = y1 / premList[j];
			matrix[(k*tid)+j]=(matrix[(k*tid)+j]+1)%2;
			printf("ligne : %i , colonne : %i , valeur : %i\n", tid , j, matrix[(k*tid)+j]);
		}
	}
	__syncthreads();
	size[tid] = sizeR;


}
