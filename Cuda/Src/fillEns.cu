#include "hip/hip_runtime.h"
/*
 * fillEns.cu
 *
 */

#include <stdio.h>
#include "header/fillEns.h"
#include "header/intList.h"
#include "header/prime.h"
#include <assert.h>
#define N 1000

__device__ hiprandState_t localState;

/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ int isInf(uint64_t *list, int size, uint64_t y){
	int i = threadIdx.x;
	int res= 0;
	__syncthreads();
	if (i < size) {
		for (int j = 0; j <= size; j++) {
			if (list[j] > y) {
				res = 1;
				break;
			}
		}
		__syncthreads();
	}
	return res;
}

/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__device__ void isBSmoothG(int *devPremList, int size, uint64_t y,int *result){
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	if(i < size) {
		uint64_t y1 = y;
		if (y1 == 0) {
			*result = 0;
		} else {

			for (int j = 0; j < size; j++) {
				while (y1 % devPremList[j] == 0) {
					y1 = y1 / devPremList[j];
				}
			}
			__syncthreads();
			if (y1 == 1) {
				*result = 1;
			} else {
				*result = 0;
			}
		}
	}
}

/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__device__ void isInEnsembleG(uint64_t *ens, uint64_t y,int size, int *res){
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	int found = 0;

	__syncthreads();
	if(i < size) {

		for (int j = 0; j < size; j++) {
			if (getValGPU(ens,j) == y) {
				found = 1;
			}
		}
	}
	__syncthreads();
	*res = found;
}

__device__ void setup_kernel ( hiprandState_t *state ) {
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init ( clock64()+id, id, 0, &state[id] );
}

__device__ void generate( hiprandState_t *globalState, uint64_t *rand, uint64_t nbr, uint64_t racN) {

	int id = threadIdx.x + blockIdx.x*blockDim.x ;
	uint64_t x;

	localState = globalState[id];
	x = (uint64_t)fmodf(hiprand(&localState),(nbr-racN)) + racN;
	globalState[id] = localState;
	rand[id] = (uint64_t) x;
}

__global__ void Generation(hiprandState_t *state,uint64_t nbr, uint64_t sqrtNBR,uint64_t *rand){
	setup_kernel(state);
}

__global__ void fillEnsR(hiprandState_t *state,Couple *R,int *size,uint64_t *Div,int sizeDiv,int * devPremList,int k,uint64_t *rand,uint64_t nbr,uint64_t sqrtNBR,char *matrix){	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	__shared__ int sizeR;
	Couple tmp;
	__shared__ char *matTmp;
	int bsmooth= -1;
	int present= -1;
	uint64_t x = 0;
	uint64_t y =  0;
	if (tid % blockDim.x == 0) {
		sizeR = 0;
		matTmp = (char *)malloc((k*k)*sizeof(char));
		memset(matTmp,0,(k*k)*sizeof(char));
	}
	__syncthreads();


	do {
		generate(state,rand,nbr,sqrtNBR);

		x = rand[tid];
		y = (x * x) % nbr;

		if (devPremList == NULL ) {
			printf("PrimeList est NULL\n");
		}
		if (k <= 0 ) {
			printf("valeur de K <= 0 \n");
		}

		isBSmoothG(devPremList, k,y,&bsmooth);

		isInEnsembleG(Div,y,sizeDiv,&present);


	} while(!bsmooth || present);

	tmp.x = x;
	tmp.y = y;
	__syncthreads();
	atomicAdd(&sizeR,1);
	uint64_t y1 = y;
	for (int j = 0; j < k; j++) {
		while (y1 % devPremList[j] == 0) {
			y1 = y1 / devPremList[j];
			matTmp[threadIdx.x*k+j] = (matTmp[threadIdx.x*k+j] + 1);
		}
	}
	__syncthreads();

	R[tid] = tmp;

	for (int j = 0; j < k; j++) {
		matrix[tid*k+j] = matTmp[threadIdx.x*k+j];
	}	
	if (tid % blockDim.x == 0) {
		free(matTmp);
		atomicAdd(size, sizeR);
	}
}
