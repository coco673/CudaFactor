#include "hip/hip_runtime.h"
/*
 * pgcd.c
 *
 *  Created on: 30 janv. 2014
 *      Author: tony
 */



#include "header/pgcd.h"
#include <string.h>

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

typedef unsigned long long uint64;

/*
 * Debut d'algorithme de Kannan-Miller-Rudolph pour obtenir le pgcd de manière parallelisée
 */
/*int pgcd(long double a, long double b){
	int n;
	if(a<b){
		n=strlen(convert(b));
	} else {
		n =strlen(convert(a));
	}

	int i = 0;
	long double *tab=malloc(n*sizeof(long double));
	switch(fork()){
		case -1 :
			exit(EXIT_FAILURE);
		case 0 :

			while(i <= n){
				tab[i] =fmodl((i*a),b);

				printf("1:%i:%s\n",i,convert(tab[i]));
				i=i+2;
			}
			break;
		default :

			i = 1;
			while(i <= n){
				tab[i] =fmodl((i*a),b);
				printf("2:%i:%s\n",i,convert(tab[i]));
				i=i+2;
			}
	}
}*/
/*
 * Change la taille de representation du binaire en size bits
 */
char *equalNBit(char *res,int size){
	int i ;
	if(strlen(res) != size){
		int n = size - strlen(res);
		char *tp = (char *)malloc(size*sizeof(char));

		for(i=0;i<n;i++){
			strcat(tp,"0");
		}
		strcat(tp,res);
		res =(char *) malloc(size*sizeof(char));
		strcpy(res,tp);
		free(tp);
	}
	return res;
}
/*
 *  Converti un entier representé sous la forme d'un long double en binaire.
 */
char *convert(long double a){
	long double n;
	long double tp[N];
	int i=0;
	n=a;


	do {
		tp[i]=fmodl(n,2);
		n=floorl(n/2);
		i++;

	} while (n>=1);


	char *res;
	res=(char *)malloc(i*sizeof(char));
	i--;

	while (i >= 0){

		char *tmp = (char *)malloc(sizeof(char));
		sprintf(tmp,"%i",(int)tp[i]);
		strcat(res,tmp);

		i--;
		free(tmp);
	}

	return res;
}

/*
 * calcule du pgcd selon l'algorithme d'euclide pour les grands entiers.
 */
int pgcd(int a,int b){

	int r,q;
	if(a<b){
		int c = a;
		a=b;
		b=c;
	}
	r = fmod(a, b);
	while (r != 0){
		if(a > b){
			a = b;
			b = r;
			r = fmod(a, b);
		}
	}
	q = b;
return q;
}

uint64 pgcdUint(uint64 u, uint64 v) {
	int shift;
	if (u == 0) {
		return v;
	}
	if (v == 0) {
		return u;
	}
	for (shift = 0; ((u | v) & 1) == 0; ++shift) {
		u >>= 1;
		v >>= 1;
	}

	while ((u & 1) == 0) {
		u >>= 1;
	}

	do {
		while ((v & 1) == 0)
			v >>= 1;
		if (u > v) {
			uint64 t = v;
			v = u;
			u = t;
		}
		v = v - u;
	} while (v != 0);

  uint64 res = u << shift;
  return res;
}
