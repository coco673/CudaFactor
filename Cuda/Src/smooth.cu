#include "hip/hip_runtime.h"
/**
 * smooth.cu
 */
#include "header/smooth.h"

int isBSmoothG(int *primeList, int size, uint64_t y) {
	int i = 0;
	if (i < size) {
		int y1 = y;
		if (y1 == 0) {
			return 0;
		} else {
			for (int j = 0; j < size;j++) {
				while (y1 % primeList[j] == 0) {
					y1 = y1/primeList[j];
				}
			}
			if (y1 == 1) {
				return 1;
			} else {
				return 0;
			}
		}
	}
	return -1;
}
