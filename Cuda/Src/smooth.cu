#include "hip/hip_runtime.h"
#include "smooth.h"

int isBSmoothG(int *primeList, int size, int y){
	int i = 0;
	if(i < size){
		int y1 = y;
		if(y1 == 0) {
			return 0;
		} else {
			for(int j = 0; j < size;j++) {
				while(y1 % primeList[j] == 0) {
					y1 = y1/primeList[j];
				}
			}
			if(y1 == 1) {
				return 1;
			} else {
				return 0;
			}
		}
	}
}
