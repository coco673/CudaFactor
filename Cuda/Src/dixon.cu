#include "hip/hip_runtime.h"
#include "header/dixon.h"
#include <unistd.h>
#include "header/fillEns.h"
#include <assert.h>

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

#define MAX_FACTORS_PER_BLOCK 20
#define NB_BLOCKS 65535
#define MAX_TEMPORARILY_FACTORS NB_BLOCKS * MAX_FACTORS_PER_BLOCK

__device__ __constant__ int devPremList[10000];

uint64_t alea(uint64_t a, uint64_t b) {
	return rand()%(b-a) +a;
}

uint64_t produitDiv(Int_List_GPU Div) {
	uint64_t res = 1;
	for (int i = 0; i < Div.Size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List_GPU Div, uint64_t val) {
	for (int i = 0; i < Div.Size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) %n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, int **matrix, int *noyau, int n) {
	int res = 1;
	int somme;
	for (int i = 0; i < sizePL; i++) {
		somme = 0;
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		}
		somme = floor(somme / 2);
		res *= (int)(pow(premList[i], somme));
		res %= n;
	}
	return res;
}

Int_List_GPU *dixon(int n) {
	//timer


	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();

	Int_List_GPU *Div = createIntList();
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		while (R->size < sizePL) {
			uint64_t x = alea(sqrt(nbr), nbr + 1);
			uint64_t y = ((uint64_t)pow(x, 2)) % nbr;

			if (isBSmoothG(premList, sizePL, y) && notIn(*Div, y) == 0) {
				tmpC.x = x;
				tmpC.y = y;
				addCouple(R, tmpC);
			}
		}
		matrix = fillMatrix(premList, sizePL, R);

		for (int i = 0; i < R->size; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);

	//timer


	return Div;
}

__host__ __device__ int **matrix1DTo2D(int *matrix, int size) {
	int **mat = new int*[size];
	for(int i = 0;i< size ; i++){
		mat[i] = new int[size];
	}
	int row = 0, col = 0;
	for (int i = 0; i < size * size; i++) {
		mat[row][col] = matrix[i];
		col = (col + 1) % size;
		if (col == 0) {
			row++;
		}
	}
	return mat;
}

Int_List_GPU *mergeDiv(Int_List_GPU *src1, Int_List_GPU *src2) {
	Int_List_GPU *result = createIntList();
	for (int i = 0; i < src1->Size; i++) {
		addInt(&result, src1->List[i]);
	}
	for (int i = 0; i < src2->Size; i++) {
		addInt(&result, src2->List[i]);
	}
	return result;
}

Int_List_GPU *factor(uint64_t n) {
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(devPremList), premList, sizePL * sizeof(int), 0, hipMemcpyHostToDevice));
	int *ptr;
	hipGetSymbolAddress((void **)&ptr, devPremList);
	Int_List_GPU *Div = createIntList();
	uint64_t nbr = n;
	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (nbr == 1) {
		return Div;
	}
	Int_List_GPU *tmpDiv;
	float racine = sqrt(nbr);
	if (ceil(racine) == racine) {
		if (Miller((int) racine, 10)) {
			addInt(&Div, (int) racine);
			addInt(&Div, (int) racine);
			return Div;
		}
		//tmpDiv = dixonGPU(nbr, n, premList, sizePL, ptr);
		tmpDiv = dixonDevice(nbr, n, premList, sizePL, ptr);
		return mergeDiv(tmpDiv, mergeDiv(Div, tmpDiv));
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	//tmpDiv = dixonGPU(nbr, n, premList, sizePL, ptr);
	tmpDiv = dixonDevice(nbr, n, premList, sizePL, ptr);
	return mergeDiv(Div, tmpDiv);
}

/*Int_List_GPU *dixonGPU(uint64_t nbr, uint64_t n, int *premList, int sizePL, int *ptr) {
	Couple_List *R = createCoupleList();
	int * sizeR = (int *) malloc(sizeof(int));
	Int_List_GPU *Div = createIntList();
	Couple *tmpC = (Couple *) malloc(sizePL * sizeof(Couple));
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	hiprandState_t *dev_state;
	Couple *dev_R;
	int *dev_sizeR;
	uint64_t *dev_Div;
	int *dev_sizeDiv;
	uint64_t *dev_rand;
	int *dev_matrix;
	int *dev_matrixMod;
	int *tmpmatrix = (int*) malloc(sizePL*sizePL * sizeof(int));
	int *tmpmatrixMod= (int*) malloc(sizePL*sizePL * sizeof(int));


	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_state,sizePL*sizeof(hiprandState_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_R,sizePL*sizeof(Couple)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeR,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeDiv,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_rand,sizePL*sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix,sizePL*sizePL*sizeof(int*)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrixMod,sizePL*sizePL*sizeof(int)));

	while (produitDiv(*Div) != nbr) {
		CUDA_CHECK_RETURN(hipMalloc((void **)&dev_Div,Div->Size*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_state,0,sizePL*sizeof(hiprandState_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_R,0,sizePL*sizeof(Couple)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeDiv,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_rand,0,sizePL*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix,0,sizePL*sizePL*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrixMod,0,sizePL*sizePL*sizeof(int)));

		CUDA_CHECK_RETURN(hipMemcpy(dev_Div,Div->List,Div->Size*sizeof(int),hipMemcpyHostToDevice));

		Generation<<<1,sizePL>>>(dev_state,nbr,(uint64_t)sqrtf(nbr),dev_rand);
		fillEnsR<<<1,sizePL>>>(dev_state,dev_R,dev_sizeR,dev_Div,Div->Size,ptr, sizePL,dev_rand,nbr,dev_matrix);
		CUDA_CHECK_RETURN(hipMemcpy(sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(hipMemcpy(tmpC,dev_R, *sizeR * sizeof(Couple),hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(hipMemcpy(tmpmatrix,dev_matrix, sizePL*sizePL*sizeof(int),hipMemcpyDeviceToHost));

		matrix = matrix1DTo2D(tmpmatrix,sizePL);
		hipDeviceSynchronize();

		for (int i = 0; i < *sizeR; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
			addCouple(R,tmpC[i]);
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;

			u = (calcul_u(*R, noyau, n));

			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));

			uint64_t pgcd1 = pgcdUint(u - v, nbr);
			uint64_t pgcd2 = pgcdUint(u + v, nbr);

			if ((pgcd1 != 1) && (pgcd1 != nbr)) {
				addInt(&Div, pgcd1);
				nbr /= pgcd1;
				while (nbr % pgcd1 == 0) {
					addInt(&Div, pgcd1);
					nbr /= pgcd1;
				}
			} else if ((pgcd2 != 1) && (pgcd2 != nbr)) {
				addInt(&Div, pgcd2);
				nbr /= pgcd2;
				while (nbr % pgcd2 == 0) {
					addInt(&Div, pgcd2);
					nbr /= pgcd2;
				}
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);
		}

		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}

		free(matrix);
		free(listNoyau);
		CUDA_CHECK_RETURN(hipFree(dev_Div));
		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}

	CUDA_CHECK_RETURN(hipFree(dev_state));
	CUDA_CHECK_RETURN(hipFree(dev_R));
	CUDA_CHECK_RETURN(hipFree(dev_sizeR));
	CUDA_CHECK_RETURN(hipFree(dev_rand));

	CUDA_CHECK_RETURN(hipFree(dev_matrix));
	CUDA_CHECK_RETURN(hipFree(dev_matrixMod));
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}*/


/*__global__ void finDixon(int *matrix, int sizePL, ) {
	//Calcul de la matrice modulo
	//possible avec 1 block et (sizePL, sizePL) threads
	int **matMod = (int **) malloc(sizePL * sizeof(int *));
	for (int i = 0; i < sizePL; i++) {
		matMod[i] = (int *) malloc(sizePL * sizeof(int));
		matMod[i] = 0;
	}
	for (int i = 0; i < sizePL; i++) {
		for (int j = 0; j < sizePL; j++) {
			matMod[i][j] = matrix[i * sizePL + j] % 2;
		}
	}

	//Gauss Jordan 1 block et 1 thread
	listeNoyau = gaussjordan_noyau(matMod, sizePL);

}*/

__device__ int **BinaryMatrix(int *matrix, int size) {
	__shared__ int **matMod;
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		matMod = (int **) malloc(size * sizeof(int *));
	}
	__syncthreads();
	if (threadIdx.x < size && threadIdx.y < size) {
		if (threadIdx.y == 0) {
			matMod[threadIdx.x] = (int *) malloc(size * sizeof(int));
		}
		__syncthreads();
		matMod[threadIdx.x][threadIdx.y] = matrix[threadIdx.x * size + threadIdx.y] % 2;
	}
	__syncthreads();
	return matMod;
}

__device__ Vector_List *gaussjordan_noyau_GPU(int **matrix, int size) {
	int pivo,jc,jl,k,j,nl,nc,l,a;
	matrix2D Mat;
	Mat.mat = matrix;
	Mat.colsNb = size;
	Mat.rowsNb = size;
	matrix2D *tmpMat = copyMatrix2D(Mat);
	nl = tmpMat->rowsNb;
	nc = tmpMat->colsNb;
	//on met des 0 sous la diagonale
	jc = 0;
	jl = 0;
	// on traite toutes les colonnes
	while (jc < nc and jl < nl) {
		//choix du pivot que l'on veut mettre en M[jl,jc]
		k = jl;
		while (tmpMat->mat[k][jc] == 0 and k < nl-1) {
			k = k + 1;
		}
		//on ne fait la suite que si on a pivo!=0
		if (tmpMat->mat[k][jc] != 0) {
			pivo = tmpMat->mat[k][jc];
			//echange de la ligne jl et de la ligne k
			//for (l = jc; l < nc; l++) {
			swapLineMatrix2D(tmpMat, jl, k);
			//}
			//fin du choix du pivot qui est M[jl,jc]
			//on met 1 sur la diagonale de la colonne jc
			for (l = 0; l < nc; l++) {
				tmpMat->mat[jl][l] = tmpMat->mat[jl][l] / pivo;
			}
			//on met des 0 au dessus de la diagonale
			// de la colonne jc
			for (k = 0; k < jl; k++) {
				a = tmpMat->mat[k][jc];
				for (l = 0; l < nc; l++) {
					tmpMat->mat[k][l] = tmpMat->mat[k][l] - tmpMat->mat[jl][l] * a;
				}
			}
			//on met des 0 sous la diag de la colonne jc
			for (k = jl + 1; k < nl; k++) {
				a = tmpMat->mat[k][jc];
				for (l = jc; l < nc; l++) {
					tmpMat->mat[k][l] = tmpMat->mat[k][l] - tmpMat->mat[jl][l] * a;
				}
			}
		}
		else{
			//on ajoute une ligne de 0 si ce n'est pas le dernier 0
			if (jl<nc-1){
				tmpMat = addLineToMatrix2D(tmpMat, 0, jl);
				nl = nl + 1;
			}
		}
		//ds tous les cas,le numero de colonne et
		//le numero de ligne augmente de 1
		jc = jc + 1;
		jl = jl + 1;
		//il faut faire toutes les colonnes
		if (jl == nl and jl < nc) {
			tmpMat = addLineToMatrix2D(tmpMat, 0, nl);
			nl++;
		}
	}
	int *noyau;
	Vector_List *listNoyau = createVectorList();
	//on enleve les lignes en trop pour avoir
	//une matrice carree de dim nc
	//on retranche la matrice identite
	//M:=M[0..nc-1]-idn(nc);
	for (int tmp = 0; tmp < nc; tmp++) {
		tmpMat->mat[tmp][tmp] = tmpMat->mat[tmp][tmp] - 1;
	}
	for(j = 0; j < nc; j++) {
		noyau = (int *) malloc(tmpMat->rowsNb * sizeof(int));
		if (tmpMat->mat[j][j] == -1) {
			for (int i = 0; i < nc; i++) {
				noyau[i] = tmpMat->mat[i][j];
			}
			addVector(listNoyau, noyau, tmpMat->rowsNb);
		}
		free(noyau);
	}
	free(tmpMat);
	return listNoyau;
}

__device__ void calculateUV(Couple *R, int *premList, int sizePL, int **matrix, int *noyau, int n, int *u, int *v) {
	if (threadIdx.x == 0) {
		int res = 1;
		for (int i = 0; i < sizePL; i++) {
			if (noyau[i] == 1) {
				res = (res * (R[i].x * R[i].x)) %n;
			}
		}
		*u = res;
	} else if (threadIdx.x == 1) {
		int res = 1;
		int somme;
		for (int i = 0; i < sizePL; i++) {
			somme = 0;
			for (int j = 1; j < sizePL; j++) {
				if (noyau[j] == 1)
					somme += matrix[j][i] * noyau[j];
			}
			somme = floor((double) somme / 2);
			res *= (int)(pow((double) premList[i], somme));
			res %= n;
		}
		*v = res;
	}
	__syncthreads();
}

__global__ void dixonParrallele(uint64_t *Div, int sizeDiv, uint64_t *newDiv, int *sizeNewDiv, int *ptr, int sizePL, uint64_t nbr) {
	__shared__ hiprandState_t *dev_state;
	__shared__ uint64_t *dev_rand;
	__shared__ Couple *dev_R;
	__shared__ int *dev_sizeR;
	__shared__ int *dev_matrix;
	__shared__ int indexCurrentBlock;
	__shared__ int u;
	__shared__ int v;
	__shared__ Vector_List *listNoyau;
	__shared__ int *noyau;
	__shared__ VEC_ELEM *tmp;
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		dev_state = (hiprandState_t *) malloc(sizePL*sizeof(hiprandState_t));
		dev_rand = (uint64_t *) malloc(sizePL*sizeof(int));
		dev_R = (Couple *) malloc(sizePL*sizeof(Couple));
		dev_sizeR = (int *) malloc(sizeof(int));
		dev_matrix = (int *) malloc(sizePL * sizePL * sizeof(int));
		indexCurrentBlock = 0;
		sizeNewDiv[blockIdx.x] = 0;
	}
	__syncthreads();
	if (threadIdx.x < sizePL && threadIdx.y == 0) {
		Generation(dev_state,nbr,(uint64_t)sqrtf(nbr),dev_rand);
		fillEnsR(dev_state,dev_R,dev_sizeR,Div,sizeDiv,ptr, sizePL,dev_rand,nbr,dev_matrix);
	}
	__syncthreads();
	int **matMod;
	int **mat;
	if (threadIdx.x < sizePL && threadIdx.y < sizePL) {
		matMod = BinaryMatrix(dev_matrix, sizePL);
	}
	__syncthreads();
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		listNoyau = gaussjordan_noyau_GPU(matMod, sizePL);
		mat = matrix1DTo2D(dev_matrix, sizePL);
	}
	__syncthreads();
	if ((threadIdx.x == 0 || threadIdx.x == 1) && threadIdx.y == 0) {
		while (listNoyau != NULL) {
			noyau = listNoyau->list->vec;
			calculateUV(dev_R, ptr, sizePL, mat, noyau, nbr, &u, &v);
			__syncthreads();
			if (threadIdx.x == 0) {

				uint64_t pgcd1 = pgcdUint(u - v, nbr);
				uint64_t pgcd2 = pgcdUint(u + v, nbr);

				if ((pgcd1 != 1) && (pgcd1 != nbr)) {
					//addInt(&Div[blockIdx.x], pgcd1);
					if (indexCurrentBlock < MAX_FACTORS_PER_BLOCK) {
						newDiv[indexCurrentBlock * blockIdx.x] = pgcd1;
						indexCurrentBlock++;
						atomicAdd(&sizeNewDiv[blockIdx.x], 1);
						nbr /= pgcd1;
						while (nbr % pgcd1 == 0) {
							//addInt(&Div[blockIdx.x], pgcd1);
							if (indexCurrentBlock < MAX_FACTORS_PER_BLOCK) {
								newDiv[indexCurrentBlock * blockIdx.x] = pgcd1;
								indexCurrentBlock++;
								atomicAdd(&sizeNewDiv[blockIdx.x], 1);
								nbr /= pgcd1;
							}
						}
					}
				} else if ((pgcd2 != 1) && (pgcd2 != nbr)) {
					//addInt(&Div[blockIdx.x], pgcd2);
					if (indexCurrentBlock < MAX_FACTORS_PER_BLOCK) {
						newDiv[indexCurrentBlock * blockIdx.x] = pgcd2;
						indexCurrentBlock++;
						atomicAdd(&sizeNewDiv[blockIdx.x], 1);
						nbr /= pgcd2;
						while (nbr % pgcd2 == 0) {
							//addInt(&Div[blockIdx.x], pgcd2);
							if (indexCurrentBlock < MAX_FACTORS_PER_BLOCK) {
								newDiv[indexCurrentBlock * blockIdx.x] = pgcd2;
								indexCurrentBlock++;
								atomicAdd(&sizeNewDiv[blockIdx.x], 1);
								nbr /= pgcd2;
							}
						}
					}
				}
				/*if (Miller(nbr, 10)) {
					addInt(&Div[blockIdx.x], nbr);
					//return Div;
				}*/
				tmp = listNoyau->list;

				listNoyau->list = listNoyau->list->suiv;
				free(tmp);
				free(noyau);
			}
		}
	}
	__syncthreads();
}

bool isIn(uint64_t *list, uint64_t elem, int size) {
	for (int i = 0; i < size; i++) {
		if (list[i] == elem) {
			return true;
		}
	}
	return false;
}

Int_List_GPU *dixonDevice(uint64_t nbr, uint64_t n, int *premList, int sizePL, int *ptr) {
	//Declarations
	Int_List_GPU *Div = createIntList(), *tmp = createIntList();
	uint64_t *tmpDiv = (uint64_t *) malloc(NB_BLOCKS * MAX_FACTORS_PER_BLOCK * sizeof(uint64_t));
	int *sizeTmpDiv = (int *) malloc(NB_BLOCKS * sizeof(int));
	dim3 threads;
	threads.x = sizePL;
	threads.y = sizePL;

	//Copies GPU
	uint64_t *dev_currentDiv;
	uint64_t *dev_nextDiv;
	int *sizeNextDiv;
	hipMalloc((int **) &dev_currentDiv, Div->Size * sizeof(int));
	hipMalloc((int **) &dev_nextDiv, NB_BLOCKS * MAX_FACTORS_PER_BLOCK * sizeof(int));
	hipMalloc((int **) &sizeNextDiv, NB_BLOCKS * sizeof(int));

	while (produitDiv(*Div) != nbr) {
		dixonParrallele<<<NB_BLOCKS, threads>>>(dev_currentDiv, Div->Size, dev_nextDiv, sizeNextDiv, ptr, sizePL, nbr);
		for (int i = 0; i < NB_BLOCKS; i++) {
			for (int j = 0; j < sizeTmpDiv[i]; j++) {
				if (!isIn(tmp->List, tmpDiv[i * MAX_FACTORS_PER_BLOCK + j], tmp->Size)) {
					addInt(&tmp, tmpDiv[i * MAX_FACTORS_PER_BLOCK + j]);
				}
			}
		}
		Div = mergeDiv(Div, tmp);
	}

	return Div;
}
