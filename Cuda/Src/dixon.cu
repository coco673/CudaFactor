#include "hip/hip_runtime.h"
#include "header/dixon.h"
#include <unistd.h>
#include "header/fillEns.h"
#include <assert.h>

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

__device__ __constant__ int devPremList[10000];

uint64_t alea(uint64_t a, uint64_t b) {
	return rand()%(b-a) +a;
}

uint64_t produitDiv(Int_List_GPU Div) {
	uint64_t res = 1;
	for (int i = 0; i < Div.Size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List_GPU Div, uint64_t val) {
	for (int i = 0; i < Div.Size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) %n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, int **matrix, int *noyau, int n) {
	int res = 1;
	int somme;
	for (int i = 0; i < sizePL; i++) {
		somme = 0;
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		}
		somme = floor(somme / 2);
		res *= (int)(pow(premList[i], somme));
		res %= n;
	}
	return res;
}

Int_List_GPU *dixon(int n) {
	//timer


	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();

	Int_List_GPU *Div = createIntList();
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		while (R->size < sizePL) {
			uint64_t x = alea(sqrt(nbr), nbr + 1);
			uint64_t y = ((uint64_t)pow(x, 2)) % nbr;

			if (isBSmoothG(premList, sizePL, y) && notIn(*Div, y) == 0) {
				tmpC.x = x;
				tmpC.y = y;
				addCouple(R, tmpC);
			}
		}
		matrix = fillMatrix(premList, sizePL, R);

		for (int i = 0; i < R->size; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);

	//timer


	return Div;
}

int **matrix1DTo2D(int *matrix, int size) {
	int **mat = new int*[size];
	for(int i = 0;i< size ; i++){
		mat[i] = new int[size];
	}
	int row = 0, col = 0;
	for (int i = 0; i < size * size; i++) {
		mat[row][col] = matrix[i];
		col = (col + 1) % size;
		if (col == 0) {
			row++;
		}
	}
	return mat;
}

Int_List_GPU *dixonGPU(uint64_t n) {
	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	printf("taille liste : %i\n", sizePL);

	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(devPremList), premList, sizePL * sizeof(int), 0, hipMemcpyHostToDevice));
	int *ptr;
	hipGetSymbolAddress((void **)&ptr, devPremList);
	Couple_List *R = createCoupleList();
	int * sizeR = (int *) malloc(sizeof(int));
	Int_List_GPU *Div = createIntList();
	Couple *tmpC = (Couple *) malloc(sizePL * sizeof(Couple));
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	uint64_t nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	hiprandState_t *dev_state;
	Couple *dev_R;
	int *dev_sizeR;
	uint64_t *dev_Div;
	int *dev_sizeDiv;
	uint64_t *dev_rand;
	int *dev_matrix;
	int *dev_matrixMod;
	int *tmpmatrix = (int*) malloc(sizePL*sizePL * sizeof(int));
	int *tmpmatrixMod= (int*) malloc(sizePL*sizePL * sizeof(int));


	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (nbr == 1) {
		return Div;
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_state,sizePL*sizeof(hiprandState_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_R,sizePL*sizeof(Couple)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeR,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeDiv,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_rand,sizePL*sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix,sizePL*sizePL*sizeof(int*)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrixMod,sizePL*sizePL*sizeof(int)));

	while (produitDiv(*Div) != nbr) {
		CUDA_CHECK_RETURN(hipMalloc((void **)&dev_Div,Div->Size*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_state,0,sizePL*sizeof(hiprandState_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_R,0,sizePL*sizeof(Couple)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeDiv,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_rand,0,sizePL*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix,0,sizePL*sizePL*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrixMod,0,sizePL*sizePL*sizeof(int)));

		CUDA_CHECK_RETURN(hipMemcpy(dev_Div,Div->List,Div->Size*sizeof(int),hipMemcpyHostToDevice));

		Generation<<<1,sizePL>>>(dev_state,nbr,(uint64_t)sqrtf(nbr),dev_rand);
		fillEnsR<<<1,sizePL>>>(dev_state,dev_R,dev_sizeR,dev_Div,Div->Size,ptr, sizePL,dev_rand,nbr,dev_matrix);
		CUDA_CHECK_RETURN(hipMemcpy(sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(hipMemcpy(tmpC,dev_R, *sizeR * sizeof(Couple),hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(hipMemcpy(tmpmatrix,dev_matrix, sizePL*sizePL*sizeof(int),hipMemcpyDeviceToHost));

		matrix = matrix1DTo2D(tmpmatrix,sizePL);
		hipDeviceSynchronize();

		for (int i = 0; i < *sizeR; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
			addCouple(R,tmpC[i]);
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;

			u = (calcul_u(*R, noyau, n));

			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));

			uint64_t pgcd1 = pgcdUint(u - v, nbr);
			uint64_t pgcd2 = pgcdUint(u + v, nbr);

			if ((pgcd1 != 1) && (pgcd1 != nbr)) {
				addInt(&Div, pgcd1);
				nbr /= pgcd1;
				while (nbr % pgcd1 == 0) {
					addInt(&Div, pgcd1);
					nbr /= pgcd1;
				}
			} else if ((pgcd2 != 1) && (pgcd2 != nbr)) {
				addInt(&Div, pgcd2);
				nbr /= pgcd2;
				while (nbr % pgcd2 == 0) {
					addInt(&Div, pgcd2);
					nbr /= pgcd2;
				}
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);
		}

		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}

		free(matrix);
		free(listNoyau);
		CUDA_CHECK_RETURN(hipFree(dev_Div));
		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}

	CUDA_CHECK_RETURN(hipFree(dev_state));
	CUDA_CHECK_RETURN(hipFree(dev_R));
	CUDA_CHECK_RETURN(hipFree(dev_sizeR));
	CUDA_CHECK_RETURN(hipFree(dev_rand));

	CUDA_CHECK_RETURN(hipFree(dev_matrix));
	CUDA_CHECK_RETURN(hipFree(dev_matrixMod));
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}
