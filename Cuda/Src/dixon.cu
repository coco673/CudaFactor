#include "hip/hip_runtime.h"
#include "dixon.h"
#include <unistd.h>
#include "fillEns.h"

int alea(int a, int b) {
	return rand()%(b-a) +a;
}

int produitDiv(Int_List Div) {
	int res = 1;
	for (int i = 0; i < Div.size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List Div, int val) {
	for (int i = 0; i < Div.size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) %n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, int **matrix, int *noyau, int n) {
	int res = 1;
	int somme = 0;
	for (int i = 0; i < sizePL; i++) {
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		} 
		somme = floor(somme / 2);
		res *= (int)(pow(premList[i], somme));
		res %= n;
	}
	return res;
}

Int_List *dixon(int n) {
	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();

	Int_List *Div = createIntList();
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(Div, nbr);
		return Div;
	}
	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		while (R->size < sizePL) {
			int x = alea(sqrt(nbr), nbr + 1);
			int y = ((int)pow(x, 2)) % nbr;

			if (isBSmoothG(premList, sizePL, y) && notIn(*Div, y) == 0) {
				tmpC.x = x;
				tmpC.y = y;
				addCouple(R, tmpC);
			}
		}
		matrix = fillMatrix(premList, sizePL, R);

		for (int i = 0; i < R->size; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}

Int_List *dixon2(int n) {
	//Declarations

	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();
	Couple *dev_R;
	Couple *Rtmp= (Couple*)malloc(sizePL*sizeof(Couple));
	Int_List *Div = createIntList();
	Int_List * dev_Div;
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	int *rand;
	int *dev_sizeR;
	int *dev_sizeDiv;
	int *dev_matrix[sizePL];
	int *dev_rand;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;
	hiprandState_t *dev_state;
	int * dev_premList;
	int sqrtNBR = (int) sqrt(n);
	hipMalloc(&dev_premList,sizePL*sizeof(int));
	hipMalloc(&dev_state,sizePL*sizeof(int));
	hipMalloc(&dev_R,sizePL*sizeof(Couple));
	hipMalloc(&dev_Div,Div->size*sizeof(Int_List));
	hipMalloc(&dev_sizeR,sizeof(int));
	hipMalloc(&dev_sizeDiv,sizeof(int));
	hipMalloc(&dev_rand,sizePL*sizeof(int));

	//Allocations

	matrixMod = (int **) malloc(sizePL * sizeof(int *));
	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
		hipMalloc(&dev_matrix[i],sizePL*sizeof(int*));
	}

	while (produitDiv(*Div) != nbr) {
		hipMemcpy(dev_Div,Div,Div->size*sizeof(Int_List),hipMemcpyHostToDevice);
		hipMemcpy(dev_sizeDiv,&(Div->size),sizeof(int),hipMemcpyHostToDevice);
		Generation<<<1,sizePL>>>(dev_state,n+1,sqrtNBR,dev_rand);
		fillEnsR<<<1,sizePL>>>(dev_state,dev_R,dev_sizeR,dev_Div,dev_sizeDiv,dev_premList,sizePL,dev_rand,nbr,dev_matrix);
		hipMemcpy(&(R->size),dev_sizeR,sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(Rtmp,dev_R,R->size * sizeof(int),hipMemcpyDeviceToHost);


		hipMemset(dev_state,0,sizePL*sizeof(hiprandState_t));
		hipMemset(dev_rand,0,sizePL*sizeof(int));
		hipMemset(dev_sizeDiv,0,sizeof(int));
		hipMemset(dev_Div,0,Div->size*sizeof(Int_List));
		hipDeviceSynchronize();

		for(int i = 0; i< R->size;i++){
			addCouple(R,Rtmp[i]);
			hipMemcpy(matrix[i],dev_matrix[i],R->size * sizeof(int *),hipMemcpyDeviceToHost);

		}
		hipDeviceSynchronize();

			listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
				break;
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
				break;
			}
			if (Miller(nbr, 10)) {
				addInt(Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(noyau);
		free(listNoyau);

		resetCoupleList(R);
	}
	hipFree(dev_state);
	hipFree(dev_rand);
	hipFree(dev_sizeDiv);
	hipFree(dev_Div);
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);
	hipFree(dev_premList);


	return Div;
}

