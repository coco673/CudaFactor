#include "hip/hip_runtime.h"
/**
 * dixon.cu
 */
 
#include "header/dixon.h"
#include <unistd.h>
#include "header/fillEns.h"
#include <assert.h>

/**
 * \def CUDA_CHECK_RETURN(value)
 * \brief Protection des appels système cuda
 */
#define CUDA_CHECK_RETURN(value) {												\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);													\
		}																		\
	}																			\

__device__ __constant__ int devPremList[10000];


int adjust_bl(int value) {
	if (value < 32) {
		return 1;
	}
	return((int) ceil(value / NB_TH_PER_BLOCK) + 1);
}

int adjust_th( int value) {
	if (value < NB_TH_PER_BLOCK) {
		return value;
	}
	return NB_TH_PER_BLOCK;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) % n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, char **matrix, int *noyau, int n) {
	int res = 1;
	int somme;
	for (int i = 0; i < sizePL; i++) {
		somme = 0;
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		}
		somme = floor(somme / 2);
		res *= (int) (pow(premList[i], somme));
		res %= n;
	}
	return res;
}

Int_List_GPU *factor(uint64_t n) {
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(devPremList), premList, sizePL * sizeof(int), 0, hipMemcpyHostToDevice));
	int *ptr;
	hipGetSymbolAddress((void **)&ptr, devPremList);
	Int_List_GPU *Div = createIntList();
	uint64_t nbr = n;
	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (nbr == 1) {
		return Div;
	}
	Int_List_GPU *tmpDiv;
	float racine = sqrt(nbr);
	if (ceil(racine) == racine) {
		if (Miller((int) racine, 10)) {
			addInt(&Div, (int) racine);
			addInt(&Div, (int) racine);
			return Div;
		}
		tmpDiv = dixonGPU(nbr, n, premList, sizePL, ptr);
		return mergeDiv(tmpDiv, mergeDiv(Div, tmpDiv));
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	tmpDiv = dixonGPU(nbr, n, premList, sizePL, ptr);
	free(premList);
	return mergeDiv(Div, tmpDiv);
}

Int_List_GPU *dixonGPU(uint64_t nbr, uint64_t n, int *premList, int sizePL, int *ptr) {
	Couple_List *R = createCoupleList();
	int sizeR = 0;
	Int_List_GPU *Div = createIntList();
	Couple *tmpC = NULL;
	char **matrix = NULL;
	char **matrixMod= NULL;
	int *noyau=NULL;
	int u, v;
	Vector_List *listNoyau = NULL;
	VEC_ELEM *tmp=NULL;

	hiprandState_t *dev_state= NULL;
	Couple *dev_R=NULL;
	int *dev_sizeR=NULL;
	uint64_t *dev_Div=NULL;
	int *dev_sizeDiv=NULL;
	uint64_t *dev_rand=NULL;
	char *dev_matrix=NULL;
	char *dev_matrixMod=NULL;
	char *tmpmatrix=NULL;


	//Allocations

	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_state,(sizePL+32)*sizeof(hiprandState_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_R,(sizePL+32)*sizeof(Couple)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeR,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeDiv,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_rand,(sizePL+32)*sizeof(uint64_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix,(sizePL+32)*(sizePL+32)*sizeof(char)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrixMod,(sizePL+32)*(sizePL+32)*sizeof(char)));
	uint64_t sqrtNBR = (uint64_t) sqrt(nbr);

	while (produitDiv(*Div) != nbr) {
		matrixMod = (char **) malloc(sizePL * sizeof(char *));

		for (int i  = 0; i < sizePL; i++) {
			matrixMod[i] = (char *) malloc(sizePL * sizeof(char));
		}
		tmpC = (Couple *) malloc(sizePL * sizeof(Couple));
		tmpmatrix = (char*) malloc((sizePL+32)*(sizePL+32) * sizeof(char));
		CUDA_CHECK_RETURN(hipMalloc((void **)&dev_Div,Div->Size*sizeof(uint64_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_state,0,(sizePL+32)*sizeof(hiprandState_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_R,0,(sizePL+32)*sizeof(Couple)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeDiv,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_rand,0,(sizePL+32)*sizeof(uint64_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix,0,(sizePL+32)*(sizePL+32)*sizeof(char)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrixMod,0,(sizePL+32)*(sizePL+32)*sizeof(char)));

		CUDA_CHECK_RETURN(hipMemcpy(dev_Div,Div->List,Div->Size*sizeof(int),hipMemcpyHostToDevice));

		Generation<<<adjust_bl(sizePL),adjust_th(sizePL)>>>(dev_state,nbr,(uint64_t)sqrt(nbr),dev_rand);
		fillEnsR<<<adjust_bl(sizePL),adjust_th(sizePL)>>>(dev_state,dev_R,dev_sizeR,dev_Div,Div->Size,ptr, sizePL,dev_rand,nbr,sqrtNBR,dev_matrix);


		CUDA_CHECK_RETURN(hipMemcpy(&sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost));
		int tp = sizeR - sizePL;
		sizeR -= tp;

		CUDA_CHECK_RETURN(hipMemcpy(tmpC,dev_R, sizeR * sizeof(Couple),hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(hipMemcpy(tmpmatrix,dev_matrix, sizePL*sizePL*sizeof(char),hipMemcpyDeviceToHost));
		matrix = matrix1DTo2D(tmpmatrix,sizePL);

		for (int i = 0; i < sizeR; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
			addCouple(R,tmpC[i]);
		}
		free(tmpmatrix);
		free(tmpC);
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);
		for (int i = 0; i < sizePL; i++) {
			free(matrixMod[i]);
		}
		free(matrixMod);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;

			u = (calcul_u(*R, noyau, n));

			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));

			uint64_t pgcd1 = pgcdRec(u - v, nbr);
			uint64_t pgcd2 = pgcdRec(u + v, nbr);

			if ((pgcd1 != 1) && (pgcd1 != nbr)) {
				addInt(&Div, pgcd1);
				nbr /= pgcd1;
				while (nbr % pgcd1 == 0) {
					addInt(&Div, pgcd1);
					nbr /= pgcd1;
				}
			} else if ((pgcd2 != 1) && (pgcd2 != nbr)) {
				addInt(&Div, pgcd2);
				nbr /= pgcd2;
				while (nbr % pgcd2 == 0) {
					addInt(&Div, pgcd2);
					nbr /= pgcd2;
				}
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}

for(int i=0;i<sizePL;i++){
		delete(matrix[i]);
}
delete(matrix);
		free(listNoyau);
		CUDA_CHECK_RETURN(hipFree(dev_Div));
		resetCoupleList(R);
	}


	CUDA_CHECK_RETURN(hipFree(dev_state));
	CUDA_CHECK_RETURN(hipFree(dev_R));
	CUDA_CHECK_RETURN(hipFree(dev_sizeR));
	CUDA_CHECK_RETURN(hipFree(dev_rand));

	CUDA_CHECK_RETURN(hipFree(dev_matrix));
	CUDA_CHECK_RETURN(hipFree(dev_matrixMod));

	free(R);
	free(premList);

	return Div;
}
