#include "hip/hip_runtime.h"
#include "dixon.h"
#include <unistd.h>
#include "fillEns.h"
#include <assert.h>
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
int alea(int a, int b) {
	return rand()%(b-a) +a;
}

int produitDiv(Int_List_GPU Div) {
	int res = 1;
	for (int i = 0; i < Div.Size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List_GPU Div, int val) {
	for (int i = 0; i < Div.Size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) %n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, int **matrix, int *noyau, int n) {
	int res = 1;
	int somme;
	for (int i = 0; i < sizePL; i++) {
		somme = 0;
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		} 
		somme = floor(somme / 2);
		res *= (int)(pow(premList[i], somme));
		res %= n;
	}
	printf("on va sortir la \n");
	return res;
}

Int_List_GPU *dixon(int n) {
	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();

	Int_List_GPU *Div = createIntList();
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		while (R->size < sizePL) {
			int x = alea(sqrt(nbr), nbr + 1);
			int y = ((int)pow(x, 2)) % nbr;

			if (isBSmoothG(premList, sizePL, y) && notIn(*Div, y) == 0) {
				tmpC.x = x;
				tmpC.y = y;
				addCouple(R, tmpC);
			}
		}
		matrix = fillMatrix(premList, sizePL, R);

		for (int i = 0; i < R->size; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}
int **matrix1DTo2D(int *matrix, int size) {
        int **mat = new int*[size];
        for(int i = 0;i< size ; i++){
        	mat[i] = new int[size];
        }
        int row = 0, col = 0;
        for (int i = 0; i < size * size; i++) {
                mat[row][col] = matrix[i];
                col = (col + 1) % size;
                if (col == 0) {
                        row++;
                }
        }
        return mat;
}

Int_List_GPU *dixon3(int n) {
	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();
	int * sizeR = (int *) malloc(sizeof(int));
	Int_List_GPU *Div = createIntList();
	Couple *tmpC = (Couple *) malloc(sizePL * sizeof(Couple));
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	hiprandState_t *dev_state;
	Couple *dev_R;
	int *dev_sizeR;
	int *dev_Div;
	int *dev_sizeDiv;
	int *dev_premList;
	int *dev_rand;
	int *dev_matrix;
	int *dev_matrixMod;
int *tmpmatrix = (int*) malloc(sizePL*sizePL * sizeof(int));
int *tmpmatrixMod= (int*) malloc(sizePL*sizePL * sizeof(int));


	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_state,sizePL*sizeof(hiprandState_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_R,sizePL*sizeof(Couple)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeR,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_premList,sizePL*sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeDiv,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_rand,sizePL*sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix,sizePL*sizePL*sizeof(int*)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrixMod,sizePL*sizePL*sizeof(int)));

		CUDA_CHECK_RETURN(hipMemcpy(dev_premList,premList,sizePL*sizeof(int),hipMemcpyHostToDevice));

	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		CUDA_CHECK_RETURN(hipMalloc((void **)&dev_Div,Div->Size*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_state,0,sizePL*sizeof(hiprandState_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_R,0,sizePL*sizeof(Couple)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR,0,sizeof(int)));
		//hipMemset(dev_Div,0,Div->Size*sizeof(Int_List_GPU));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeDiv,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_rand,0,sizePL*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix,0,sizePL*sizePL*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrixMod,0,sizePL*sizePL*sizeof(int)));

		CUDA_CHECK_RETURN(hipMemcpy(dev_Div,Div->List,Div->Size*sizeof(int),hipMemcpyHostToDevice));

		Generation<<<1,sizePL>>>(dev_state,nbr,(int)sqrtf(nbr),dev_rand);

		fillEnsR<<<1,sizePL>>>(dev_state,dev_R,dev_sizeR,dev_Div,Div->Size,dev_premList,sizePL,dev_rand,nbr,dev_matrix);

		CUDA_CHECK_RETURN(hipMemcpy(sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost));
		printf("la taille de R %i\n",*sizeR);
*sizeR=*sizeR-1;
		CUDA_CHECK_RETURN(hipMemcpy(tmpC,dev_R, *sizeR * sizeof(Couple),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(tmpmatrix,dev_matrix, sizePL*sizePL*sizeof(int),hipMemcpyDeviceToHost));

		matrix = matrix1DTo2D(tmpmatrix,sizePL);
		hipDeviceSynchronize();

		for (int i = 0; i < *sizeR; i++) {
			for (int j = 0; j < sizePL; j++) {
				addCouple(R,tmpC[j]);
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		printf("on arrive à gauss\n");
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);
printf("ok gauss\n");
int nbX= 0;
		while (listNoyau->list != NULL) {
			printf("valeur nbX %i\n",nbX);
			noyau = listNoyau->list->vec;
			printf("calcul de u\n");

			printf("*-------------------------------------------*\n");
			u = (calcul_u(*R, noyau, n));
			printf("good U\n");
printf("size PL %i\n",sizePL);
			for(int i = 0; i< sizePL ; i++){
				printf("%i:: %i \n",noyau[i],i);
			}
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));

			printf("good V\n");
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);
nbX++;
		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);
		hipFree(dev_Div);
		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	hipFree(dev_state);
	hipFree(dev_R);
	hipFree(dev_sizeR);
	//hipFree(dev_Div);
	hipFree(dev_rand);

	hipFree(dev_matrix);
	hipFree(dev_matrixMod);
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}
