#include "hip/hip_runtime.h"
#include "header/dixon.h"
#include <unistd.h>
#include "header/fillEns.h"
#include <assert.h>

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

__device__ __constant__ int devPremList[10000];


int adjust_bl(int value){
	if(value < 32){
		return 1;
	}
	return(ceil(value/NB_TH_PER_BLOCK)+1);
}

int adjust_th( int value){
	if(value < NB_TH_PER_BLOCK){
		return value;
	}
	return NB_TH_PER_BLOCK;
}
uint64_t alea(uint64_t a, uint64_t b) {
	return rand()%(b-a) +a;
}

uint64_t produitDiv(Int_List_GPU Div) {
	uint64_t res = 1;
	for (int i = 0; i < Div.Size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List_GPU Div, uint64_t val) {
	for (int i = 0; i < Div.Size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) %n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, char **matrix, int *noyau, int n) {
	int res = 1;
	int somme;
	for (int i = 0; i < sizePL; i++) {
		somme = 0;
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		}
		somme = floor(somme / 2);
		res *= (int)(pow(premList[i], somme));
		res %= n;
	}
	return res;
}

/*Int_List_GPU *dixon(int n) {
	//timer


	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();

	Int_List_GPU *Div = createIntList();
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		while (R->size < sizePL) {
			uint64_t x = alea(sqrt(nbr), nbr + 1);
			uint64_t y = ((uint64_t)pow(x, 2)) % nbr;

			if (isBSmoothG(premList, sizePL, y) && notIn(*Div, y) == 0) {
				tmpC.x = x;
				tmpC.y = y;
				addCouple(R, tmpC);
			}
		}
		matrix = fillMatrix(premList, sizePL, R);

		for (int i = 0; i < R->size; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(&Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);

	//timer


	return Div;
}*/

char **matrix1DTo2D(char *matrix, int size) {
	char **mat = new char*[size];
	for(int i = 0;i< size ; i++){
		mat[i] = new char[size];
	}
	int row = 0, col = 0;
	for (int i = 0; i < size * size; i++) {
		mat[row][col] = matrix[i];
		col = (col + 1) % size;
		if (col == 0) {
			row++;
		}
	}
	return mat;
}

Int_List_GPU *mergeDiv(Int_List_GPU *src1, Int_List_GPU *src2) {
	Int_List_GPU *result = createIntList();
	for (int i = 0; i < src1->Size; i++) {
		addInt(&result, src1->List[i]);
	}
	for (int i = 0; i < src2->Size; i++) {
		addInt(&result, src2->List[i]);
	}
	delete[](src1);
	delete[](src2);
	return result;
}

Int_List_GPU *factor(uint64_t n) {
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	printf("size pl = %i\n",sizePL);
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(devPremList), premList, sizePL * sizeof(int), 0, hipMemcpyHostToDevice));
	int *ptr;
	hipGetSymbolAddress((void **)&ptr, devPremList);
	Int_List_GPU *Div = createIntList();
	uint64_t nbr = n;
	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(&Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (nbr == 1) {
		return Div;
	}
	Int_List_GPU *tmpDiv;
	float racine = sqrt(nbr);
	if (ceil(racine) == racine) {
		if (Miller((int) racine, 10)) {
			addInt(&Div, (int) racine);
			addInt(&Div, (int) racine);
			return Div;
		}
		tmpDiv = dixonGPU(nbr, n, premList, sizePL, ptr);
		return mergeDiv(tmpDiv, mergeDiv(Div, tmpDiv));
	}
	if (Miller(nbr, 10)) {
		addInt(&Div, nbr);
		return Div;
	}
	tmpDiv = dixonGPU(nbr, n, premList, sizePL, ptr);
	free(premList);
	return mergeDiv(Div, tmpDiv);
}

Int_List_GPU *dixonGPU(uint64_t nbr, uint64_t n, int *premList, int sizePL, int *ptr) {
	Couple_List *R = createCoupleList();
	//int * sizeR = (int *) malloc(sizeof(int));
	int sizeR = 0;
	Int_List_GPU *Div = createIntList();
	Couple *tmpC = NULL;
	char **matrix = NULL;
	char **matrixMod= NULL;
	int *noyau=NULL;
	int u, v;
	Vector_List *listNoyau = NULL;
	VEC_ELEM *tmp=NULL;

	hiprandState_t *dev_state= NULL;
	Couple *dev_R=NULL;
	int *dev_sizeR=NULL;
	uint64_t *dev_Div=NULL;
	int *dev_sizeDiv=NULL;
	uint64_t *dev_rand=NULL;
	char *dev_matrix=NULL;
	char *dev_matrixMod=NULL;
	char *tmpmatrix=NULL;


	//Allocations

	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_state,(sizePL+32)*sizeof(hiprandState_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_R,(sizePL+32)*sizeof(Couple)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeR,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_sizeDiv,sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_rand,(sizePL+32)*sizeof(uint64_t)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrix,(sizePL+32)*(sizePL+32)*sizeof(char)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_matrixMod,(sizePL+32)*(sizePL+32)*sizeof(char)));

	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		matrixMod = (char **) malloc(sizePL * sizeof(char *));

		for (int i  = 0; i < sizePL; i++) {
			matrixMod[i] = (char *) malloc(sizePL * sizeof(char));
		}
		tmpC = (Couple *) malloc(sizePL * sizeof(Couple));
		tmpmatrix = (char*) malloc((sizePL+32)*(sizePL+32) * sizeof(char));
		CUDA_CHECK_RETURN(hipMalloc((void **)&dev_Div,Div->Size*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_state,0,(sizePL+32)*sizeof(hiprandState_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_R,0,(sizePL+32)*sizeof(Couple)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeDiv,0,sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_rand,0,(sizePL+32)*sizeof(uint64_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix,0,(sizePL+32)*(sizePL+32)*sizeof(char)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrixMod,0,(sizePL+32)*(sizePL+32)*sizeof(char)));

		CUDA_CHECK_RETURN(hipMemcpy(dev_Div,Div->List,Div->Size*sizeof(int),hipMemcpyHostToDevice));
		Generation<<<adjust_bl(sizePL),adjust_th(sizePL)>>>(dev_state,nbr,(uint64_t)sqrtf(nbr),dev_rand);
		fillEnsR<<<adjust_bl(sizePL),adjust_th(sizePL)>>>(dev_state,dev_R,dev_sizeR,dev_Div,Div->Size,ptr, sizePL,dev_rand,nbr,dev_matrix);


		CUDA_CHECK_RETURN(hipMemcpy(&sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost));
		int tp = sizeR - sizePL;
		sizeR -= tp;
		CUDA_CHECK_RETURN(hipMemcpy(tmpC,dev_R, sizeR * sizeof(Couple),hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(hipMemcpy(tmpmatrix,dev_matrix, sizePL*sizePL*sizeof(char),hipMemcpyDeviceToHost));
		matrix = matrix1DTo2D(tmpmatrix,sizePL);
		for (int i = 0; i < sizeR; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
			addCouple(R,tmpC[i]);
		}
		free(tmpmatrix);
		free(tmpC);
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);
		for (int i = 0; i < sizePL; i++) {
					free(matrixMod[i]);
				}
				free(matrixMod);

				while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;

			u = (calcul_u(*R, noyau, n));

			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));

			uint64_t pgcd1 = pgcdUint(u - v, nbr);
			uint64_t pgcd2 = pgcdUint(u + v, nbr);

			if ((pgcd1 != 1) && (pgcd1 != nbr)) {
				addInt(&Div, pgcd1);
				nbr /= pgcd1;
				while (nbr % pgcd1 == 0) {
					addInt(&Div, pgcd1);
					nbr /= pgcd1;
				}
			} else if ((pgcd2 != 1) && (pgcd2 != nbr)) {
				addInt(&Div, pgcd2);
				nbr /= pgcd2;
				while (nbr % pgcd2 == 0) {
					addInt(&Div, pgcd2);
					nbr /= pgcd2;
				}
			}
			if (Miller(nbr, 10)) {
				addInt(&Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}


		delete[](matrix);
		free(listNoyau);
		CUDA_CHECK_RETURN(hipFree(dev_Div));
		resetCoupleList(R);
		printf("c est pas fini\n");
	}


	CUDA_CHECK_RETURN(hipFree(dev_state));
	CUDA_CHECK_RETURN(hipFree(dev_R));
	CUDA_CHECK_RETURN(hipFree(dev_sizeR));
	CUDA_CHECK_RETURN(hipFree(dev_rand));

	CUDA_CHECK_RETURN(hipFree(dev_matrix));
	CUDA_CHECK_RETURN(hipFree(dev_matrixMod));

	free(R);
	//free(sizeR);
	free(premList);

	return Div;
}
