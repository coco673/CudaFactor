#include "hip/hip_runtime.h"
#include "dixon.h"
#include <unistd.h>
#include "fillEns.h"

int alea(int a, int b) {
	return rand()%(b-a) +a;
}

int produitDiv(Int_List Div) {
	int res = 1;
	for (int i = 0; i < Div.size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List Div, int val) {
	for (int i = 0; i < Div.size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

int calcul_u(Couple_List R, int *noyau, int n) {
	int res = 1;
	for (int i = 0; i < R.size; i++) {
		if (noyau[i] == 1) {
			res = (res * (getCouple(R, i).x * getCouple(R, i).x)) %n;
		}
	}
	return res;
}

int calcul_v(int *premList, int sizePL, Couple_List R, int **matrix, int *noyau, int n) {
	int res = 1;
	int somme = 0;
	for (int i = 0; i < sizePL; i++) {
		for (int j = 1; j < R.size; j++) {
			if (noyau[j] == 1)
				somme += matrix[j][i] * noyau[j];
		} 
		somme = floor(somme / 2);
		res *= (int)(pow(premList[i], somme));
		res %= n;
	}
	return res;
}

Int_List *dixon(int n) {
	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();

	Int_List *Div = createIntList();
	Couple tmpC;
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(Div, nbr);
		return Div;
	}
	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {
		while (R->size < sizePL) {
			int x = alea(sqrt(nbr), nbr + 1);
			int y = ((int)pow(x, 2)) % nbr;

			if (isBSmoothG(premList, sizePL, y) && notIn(*Div, y) == 0) {
				tmpC.x = x;
				tmpC.y = y;
				addCouple(R, tmpC);
			}
		}
		matrix = fillMatrix(premList, sizePL, R);

		for (int i = 0; i < R->size; i++) {
			for (int j = 0; j < sizePL; j++) {
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}
		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}


Int_List *dixon3(int n) {
	//Declarations

	//int borne = sqrt(exp(sqrt(log(n)*log(log(n)))));
	int borne = ceil(sqrt(exp(sqrt(2 * log(n) * log(log(n))))));
	int sizePL;
	int *premList = generatePrimeList(borne, &sizePL);
	Couple_List *R = createCoupleList();
	int * sizeR = (int *) malloc(sizeof(int));
	Int_List *Div = createIntList();
	Couple *tmpC = (Couple *) malloc(sizePL * sizeof(Couple));
	int **matrix;
	int **matrixMod;
	int *noyau;
	int u, v;
	int nbr = n;
	Vector_List *listNoyau;
	VEC_ELEM *tmp;

	hiprandState_t *dev_state;
	Couple *dev_R;
	int *dev_sizeR;
	Int_List *dev_Div;
	int *dev_sizeDiv;
	int *dev_premList;
	int *dev_rand;
	int *dev_matrix[sizePL];
	int *dev_matrixMod[sizePL];

	//Allocations
	matrixMod = (int **) malloc(sizePL * sizeof(int *));

	for (int i  = 0; i < sizePL; i++) {
		matrixMod[i] = (int *) malloc(sizePL * sizeof(int));
	}

	int index = 0;
	while(index < sizePL) {
		if (nbr % premList[index] == 0) {
			addInt(Div, premList[index]);
			nbr /= premList[index];
		} else {
			index++;
		}
	}
	if (Miller(nbr, 10)) {
		addInt(Div, nbr);
		return Div;
	}
	hipMalloc((void **)&dev_state,sizePL*sizeof(hiprandState_t));
	hipMalloc((void **)&dev_R,sizePL*sizeof(Couple));
	hipMalloc((void **)&dev_sizeR,sizeof(int));
	hipMalloc((void **)&dev_Div,Div->size*sizeof(Int_List));
	hipMalloc((void **)&dev_sizeDiv,sizeof(int));
	hipMalloc((void **)&dev_rand,sizePL*sizeof(int));
	for(int i = 0; i < sizePL; i++){
		hipMalloc((void **)&dev_matrix[i],sizePL*sizeof(int));
		hipMalloc((void **)&dev_matrixMod[i],sizePL*sizeof(int));
	}

	printf("entree dans Dixon\n");
	while (produitDiv(*Div) != nbr) {

		hipMemset(dev_state,0,sizePL*sizeof(hiprandState_t));
		hipMemset(dev_R,0,sizePL*sizeof(Couple));
		hipMemset(dev_sizeR,0,sizeof(int));
		hipMemset(dev_Div,0,Div->size*sizeof(Int_List));
		hipMemset(dev_sizeDiv,0,sizeof(int));
		hipMemset(dev_rand,0,sizePL*sizeof(int));
		for(int i = 0; i < sizePL; i++){
			hipMemset(dev_matrix[i],0,sizePL*sizeof(int));
			hipMemset(dev_matrixMod[i],0,sizePL*sizeof(int));
		}
		hipMemcpy(dev_Div,Div,Div->size*sizeof(Int_List),hipMemcpyHostToDevice);

		Generation<<<1,sizePL>>>(dev_state,nbr,(int)sqrtf(nbr),dev_rand);

		fillEnsR<<<1,sizePL>>>(dev_state,dev_R,dev_sizeR,dev_Div,&(dev_Div->size),dev_premList,sizePL,dev_rand,nbr,dev_matrix);

		hipMemcpy(sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost);
		printf("la taille de R %i\n",*sizeR);
		hipMemcpy(tmpC,dev_R, *sizeR * sizeof(Couple),hipMemcpyDeviceToHost);
		hipMemcpy(matrix,dev_matrix, sizePL*sizeof(int),hipMemcpyDeviceToHost);
		for(int i =0 ; i < sizePL; i++){
			hipMemcpy(matrix[i],dev_matrix[i],sizePL*sizeof(int),hipMemcpyDeviceToHost);
			printf("%i :: %i\n",tmpC[i].x,tmpC[i].y);
		}
		hipDeviceSynchronize();

		for (int i = 0; i < *sizeR; i++) {
			for (int j = 0; j < sizePL; j++) {
				addCouple(R,tmpC[j]);
				printf("%i :: %i\n",tmpC[i].x,tmpC[i].y);
				matrixMod[i][j] = matrix[i][j] % 2;
			}
		}

		listNoyau = gaussjordan_noyau(matrixMod, sizePL);

		while (listNoyau->list != NULL) {
			noyau = listNoyau->list->vec;
			u = (calcul_u(*R, noyau, n));
			v = (calcul_v(premList, sizePL, *R, matrix, noyau,n));
			if ((pgcdUint(u - v, nbr) != 1) && (pgcdUint(u - v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u - v, nbr));
				nbr /= pgcdUint(u - v, nbr);
			} else if ((pgcdUint(u + v, nbr) != 1) && (pgcdUint(u + v, nbr) != nbr)) {
				addInt(Div, pgcdUint(u + v, nbr));
				nbr /= pgcdUint(u + v, nbr);
			}
			if (Miller(nbr, 10)) {
				addInt(Div, nbr);
				return Div;
			}
			tmp = listNoyau->list;

			listNoyau->list = listNoyau->list->suiv;
			free(tmp);
			free(noyau);

		}
		for (int i = 0; i < sizePL; i++) {
			free(matrix[i]);
		}
		free(matrix);
		free(listNoyau);

		resetCoupleList(R);
	}
	for (int i = 0; i < sizePL; i++) {
		free(matrixMod[i]);
	}
	hipFree(dev_state);
	hipFree(dev_R);
	hipFree(dev_sizeR);
	hipFree(dev_Div);
	hipFree(dev_rand);
	for(int i = 0; i < sizePL; i++){
		hipFree(dev_matrix[i]);
		hipFree(dev_matrixMod[i]);
	}
	hipFree(dev_matrix);
	hipFree(dev_matrixMod);
	free(matrixMod);
	free(R);
	free(premList);

	return Div;
}
