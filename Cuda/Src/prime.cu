#include "hip/hip_runtime.h"
#include "header/prime.h"
/*
// Cette fonctions elimine les multiples de chaques nombres
// Ce qui a pour effet d'enlever les nombres non premiers
// list contient tous les nombres de 2 a borne
// borne est la borne
__global__ void eratosthene(int *list, int borne) {
	int id = blockIdx.x;
	int limite = (int) sqrt((double) borne);
	for (int i = id + 1; i <= borne; i++) {
		if (list[i] != 0 && (list[i] % list[id]) == 0) {
			list[i] = 0;
		}
	}
}

// Cette fonction remplie la liste des nombres de 2 à la borne choisie (nb de blocks cuda)
// list est la liste a remplir
__global__ void listNumbers(int *list) {
	int id = blockIdx.x;
	if (id != 0 && id != 1) {
		list[id - 2] = id;
	}
}

// Copie d'un tableau en Cuda
// src est le tableau source
// dest est le tableau de destination
// size est la taille de src
__global__ void copyTab(int *src, int *dest, int size) {
	int id = blockIdx.x;
	if (id < size) {
		dest[id] = src[id];
	}
}

// Cette fonction reduit le tableau des nombres premiers en enlevant les 0 inutiles
// list est le tableau de premiers avec les zeros inutiles
// result est le tableau contenant le resultat
// borne est la borne choisie
// Cette fonction retourne la taille de result
int primeList(int *list, int *result, int borne) {
	int id = 0;
	int res;
	int *dev_res;
	int *dev_list;
	for (int i = 0; i < borne; i++) {
		if (list[i] != 0) {
			result[id] = list[i];
			id++;
		}
	}
	res = hipMalloc(&dev_list, id);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	res = hipMalloc(&dev_res, id);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	res = hipMemcpy(dev_list, result, id, hipMemcpyHostToDevice);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	copyTab<<<id, 1>>>(dev_list, dev_res, id);
	res = hipMemcpy(result, dev_res, id, hipMemcpyDeviceToHost);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	hipFree(dev_list);
	hipFree(dev_res);
	return id;
}

// Genere la liste d'entiers premiers
// borne est la limite max des nb premiers generes
// tailleResult est la taille du tableau retourne
// Cette fonction retourne le tableau d'entiers
int *generatePrimeList(int borne, int *tailleResult) {
	if (borne < 2) {
		return NULL;
	}
	int *numbers;
	int *dev_numbers;
	//tableaux des nombres de 2 a borne
	numbers = (int *) malloc((borne - 1) * sizeof(int));
	int ret = hipMalloc(&dev_numbers, ((borne - 1) * sizeof(int)));
	if (ret != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	//borne + 1 pour inclure la borne les blocks rentrent leur id
	listNumbers<<<borne + 1, 1>>>(dev_numbers);
	eratosthene<<<borne, 1>>>(dev_numbers, borne);
	hipMemcpy(numbers, dev_numbers, (borne - 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (ret != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	int *result = (int *) malloc(borne * sizeof(int));
	int taille = primeList(numbers, result, borne);
	*tailleResult = taille;
	hipFree(dev_numbers);
	free(numbers);
	return result;
}
*/

__global__ void fillList(int *list, int borne) {
	int id = blockIdx.x;
	if (id > 1 && id <= borne) {
		list[id - 2] = id;
	}
}

__global__ void eratosthene(int *list) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if (tid > 1 && tid != list[bid]) {
		if (list[bid] % tid == 0) {
			list[bid] = 0;
		}
	}
}

int *generatePrimeList(int borne, int *size) {
	int *list = (int *) malloc((borne - 1) * sizeof(int));
	int *dev_list;
	hipMalloc((int **)&dev_list, (borne - 1) * sizeof(int));
	fillList<<<borne + 1, 1>>>(dev_list, borne);
	eratosthene<<<borne - 1, borne - 1>>>(dev_list);
	hipMemcpy(list, dev_list, (borne - 1) * sizeof(int), hipMemcpyDeviceToHost);
	*size = 0;
	for (int i = 0; i < (borne - 1); i++) {
		if (list[i] != 0) {
			(*size)++;
		}
	}
	int *res = (int *) malloc((*size) * sizeof(int));
	for (int i = 0, j = 0; i < (borne - 1); i++) {
		if (list[i] != 0) {
			res[j] = list[i];
			j++;
		}
	}
	hipFree(dev_list);
	free(list);
	return res;
}
