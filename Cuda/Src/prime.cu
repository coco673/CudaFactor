#include "hip/hip_runtime.h"
/**
 * prime.cu
 */
#include "header/prime.h"

__global__ void fillList(int *list, uint64_t borne) {
	int id = blockIdx.x;
	if (id > 1 && id <= borne) {
		list[id - 2] = id;
	}
}

__global__ void eratosthene(int *list) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if (tid > 1 && tid != list[bid]) {
		if (list[bid] % tid == 0) {
			list[bid] = 0;
		}
	}
}

int *generatePrimeList(int borne, int *size) {
	int *list = (int *) malloc((borne - 1) * sizeof(int));
	int *dev_list;
	int cures = hipMalloc((int **)&dev_list, (borne - 1) * sizeof(int));
	if (cures != hipSuccess) {
		fprintf(stderr, "1 ; %s", hipGetErrorString(hipGetLastError()));
		exit(1);
	}
	fillList<<<borne + 1, 1>>>(dev_list, borne);
	eratosthene<<<borne - 1, borne - 1>>>(dev_list);
	cures = hipMemcpy(list, dev_list, (borne - 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (cures != hipSuccess) {
		fprintf(stderr, "2 ; %s", hipGetErrorString(hipGetLastError()));
		exit(1);
	}
	*size = 0;
	for (int i = 0; i < (borne - 1); i++) {
		if (list[i] != 0) {
			(*size)++;
		}
	}
	int *res = (int *) malloc((*size) * sizeof(int));
	for (int i = 0, j = 0; i < (borne - 1); i++) {
		if (list[i] != 0) {
			res[j] = list[i];
			j++;
		}
	}
	cures = hipFree(dev_list);
	if (cures != hipSuccess) {
		fprintf(stderr, "3 ; %s", hipGetErrorString(hipGetLastError()));
		exit(1);
	}
	free(list);
	return res;
}
