#include "hip/hip_runtime.h"
#include "prime.h"
/*
// Cette fonctions elimine les multiples de chaques nombres
// Ce qui a pour effet d'enlever les nombres non premiers
// list contient tous les nombres de 2 a borne
// borne est la borne
__global__ void eratosthene(int *list, int borne) {
	int id = blockIdx.x;
	int limite = (int) sqrt((double) borne);
	for (int i = id + 1; i <= borne; i++) {
		if (list[i] != 0 && (list[i] % list[id]) == 0) {
			list[i] = 0;
		}
	}
}

// Cette fonction remplie la liste des nombres de 2 à la borne choisie (nb de blocks cuda)
// list est la liste a remplir
__global__ void listNumbers(int *list) {
	int id = blockIdx.x;
	if (id != 0 && id != 1) {
		list[id - 2] = id;
	}
}

// Copie d'un tableau en Cuda
// src est le tableau source
// dest est le tableau de destination
// size est la taille de src
__global__ void copyTab(int *src, int *dest, int size) {
	int id = blockIdx.x;
	if (id < size) {
		dest[id] = src[id];
	}
}

// Cette fonction reduit le tableau des nombres premiers en enlevant les 0 inutiles
// list est le tableau de premiers avec les zeros inutiles
// result est le tableau contenant le resultat
// borne est la borne choisie
// Cette fonction retourne la taille de result
int primeList(int *list, int *result, int borne) {
	int id = 0;
	int res;
	int *dev_res;
	int *dev_list;
	for (int i = 0; i < borne; i++) {
		if (list[i] != 0) {
			result[id] = list[i];
			id++;
		}
	}
	res = hipMalloc(&dev_list, id);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	res = hipMalloc(&dev_res, id);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	res = hipMemcpy(dev_list, result, id, hipMemcpyHostToDevice);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	copyTab<<<id, 1>>>(dev_list, dev_res, id);
	res = hipMemcpy(result, dev_res, id, hipMemcpyDeviceToHost);
	if (res != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	hipFree(dev_list);
	hipFree(dev_res);
	return id;
}

// Genere la liste d'entiers premiers
// borne est la limite max des nb premiers generes
// tailleResult est la taille du tableau retourne
// Cette fonction retourne le tableau d'entiers
int *generatePrimeList(int borne, int *tailleResult) {
	if (borne < 2) {
		return NULL;
	}
	int *numbers;
	int *dev_numbers;
	//tableaux des nombres de 2 a borne
	numbers = (int *) malloc((borne - 1) * sizeof(int));
	int ret = hipMalloc(&dev_numbers, ((borne - 1) * sizeof(int)));
	if (ret != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	//borne + 1 pour inclure la borne les blocks rentrent leur id
	listNumbers<<<borne + 1, 1>>>(dev_numbers);
	eratosthene<<<borne, 1>>>(dev_numbers, borne);
	hipMemcpy(numbers, dev_numbers, (borne - 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (ret != hipSuccess) {
		fprintf(stderr, "%s", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	int *result = (int *) malloc(borne * sizeof(int));
	int taille = primeList(numbers, result, borne);
	*tailleResult = taille;
	hipFree(dev_numbers);
	free(numbers);
	return result;
}
*/

__global__ void fillList(int *list, uint64_t borne) {
	int id = blockIdx.x;
	if (id > 1 && id <= borne) {
		list[id - 2] = id;
	}
}

__global__ void eratosthene(int *list) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if (tid > 1 && tid != list[bid]) {
		if (list[bid] % tid == 0) {
			list[bid] = 0;
		}
	}
}

uint64_t *generatePrimeList(uint64_t borne, int *size) {
	int *list = (int *) malloc((borne - 1) * sizeof(int));
	int *dev_list;
	int cures = hipMalloc((int **)&dev_list, (borne - 1) * sizeof(int));
	if (cures != hipSuccess) {
		fprintf(stderr, "1 ; %s", hipGetErrorString(hipGetLastError()));
		exit(1);
	}
	fillList<<<borne + 1, 1>>>(dev_list, borne);
	eratosthene<<<borne - 1, borne - 1>>>(dev_list);
	cures = hipMemcpy(list, dev_list, (borne - 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (cures != hipSuccess) {
		fprintf(stderr, "2 ; %s", hipGetErrorString(hipGetLastError()));
		exit(1);
	}
	*size = 0;
	for (uint64_t i = 0; i < (borne - 1); i++) {
		if (list[i] != 0) {
			(*size)++;
		}
	}
	uint64_t *res = (uint64_t *) malloc((*size) * sizeof(uint64_t));
	for (int i = 0, j = 0; i < (borne - 1); i++) {
		if (list[i] != 0) {
			res[j] = list[i];
			j++;
		}
	}
	cures = hipFree(dev_list);
	if (cures != hipSuccess) {
		fprintf(stderr, "3 ; %s", hipGetErrorString(hipGetLastError()));
		exit(1);
	}
	free(list);
	return res;
}
