#include "hip/hip_runtime.h"
/*
 * fillEnsemble.cu
 *
 * Created on: 5 Feb. 2014
 * 	  Autor : tony
 */

#include "header/fillEnsemble.h"
#include "header/prime.h"
#include <string.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//nombre de bits à générer
#define N 1000

/**
 * retourne true si l'entier est B-friable false sinon. (Version cpu)
 */
bool isBSmooth(int *list,int size, int y){

	int i =0;
	bool val = true;
	if(list == NULL){
		fprintf(stderr,"list null\n");
		return -1;
	}
	while(i < size){
		if(list[i] > y){
			return false;
		}
		i++;
	}
	return val;
}
/**
 * Retourne true si l'entier fait partie de l'ensemble false sinon
 * (version CPu)
 */
bool isInEnsemble(ensemble ens, int y,int size){
	int i =0;
	if(ens == NULL){
		return false;
	}
	while(i <size ){
		if(ens[i].ind.val == y){
			return true;
		}
		i++;
	}
	return false;
}

/**
 * Mode GPU.
 * Verifie si chaque valeur contenue dans la liste des premiers (de taille size)
 * est inférieur a la valeur y.
 */
__device__ int isInf(int *list, int size, int y){
	int i = threadIdx.x;
	volatile __shared__ int res;
	if(i == 0){
		res = 0;
	}
	__syncthreads();
	if(i < size){

		if(list[i] > y){
			res=1;
		}
		__syncthreads();
	}
	return res;
}

/**
 * Mode GPU
 * Verifie si la valeur y est B-friable.
 * list est la liste des premiers de la borne
 * result le resultat retourné
 */
__device__ void isBSmoothG(int *list,int size, int y,int *result){
	int i =threadIdx.x;
	volatile __shared__ int found;
	if(threadIdx.x == 0) found = 0;
	__syncthreads();

	if(found== 0 && i < size){
		bool inf = isInf(list,size,y);
		if(!inf){
			found = 1;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

/**
 * Mode GPU
 * Verifie si la valeur y fait partie de l'ensemble ens et stocke le resultat
 * res.
 * size est la taille de l'ensemble.
 */

__device__ void isInEnsembleG(ensemble ens, int y,int size, int *res){
	int i = threadIdx.x;
	volatile __shared__ int found;

	if (i == 0){
		found =0;
	}

	if(i < size){
		__syncthreads();
		if(ens == NULL || ens[i].ind.val == y){
			found = 1;
			*res = found;
		}
		__syncthreads();
		*res = found;

	}else if(size == 0){
		*res= found;
	}
}

/**
 * Construit l'ensemble R. (version CPU)
 */
void fillEnsemble(ensemble r,int nbr,int borne,ensemble div
		,int sizeDiv){
	int m=0;
	int k;
	int *p = generatePrimeList(borne,&k);
	k--;
	r = initEns(&m);

	int x;
	int y;
	int racN=sqrt(nbr);

	srand(time(NULL));

	while(m < k+1){
		x = racN+(rand() % ((nbr-1) - racN));
		y = pow(x,2);
		y=y%nbr;

		if(isBSmooth(p,k,y) && !isInEnsemble(div,y,sizeDiv)){
			addCouple(&r,x,y,&m);

		}
	}
}

__device__ void setup_kernel ( hiprandState_t *state )
{
	//int id = threadIdx.x + blockIdx.x+ blockDim.x;
	int id = threadIdx.x + blockIdx.x;
	hiprand_init ( clock64()+id, id, 0, &state[id] );
}

__device__ void generate( hiprandState_t *globalState, int *rand, int nbr, int racN)
{
	//int id = threadIdx.x + blockIdx.x+ blockDim.x;
	int id = threadIdx.x + blockIdx.x;
	float x;

	hiprandState_t localState = globalState[id];
	for(int n = 0; n < N; n++) {
		x = fmodf(hiprand(&localState),(nbr-racN)) + racN;
	}
	globalState[id] = localState;
	rand[id] = (int) x;
}


__device__ int generateRonce(ensemble r,int *p,int k,int nbr,ensemble div,int sizeDiv, int *sizeR,int *matrix){
	int i =  threadIdx.x;
	uint x;
	uint y;

	volatile __shared__ int ret;
	if (i == 0) {
		ret = -1;
	}
	__syncthreads();

	int racN = (int)sqrtf(nbr);

	int *rand=(int*)malloc((blockDim.x*gridDim.x)*sizeof(int));
	int *bsmooth = (int *)malloc(sizeof(int));
	int *present = (int *)malloc(sizeof(int));
	hiprandState_t *devStates;
	devStates=(hiprandState_t *) malloc ((blockDim.x*gridDim.x)*sizeof( hiprandState_t ) );

	// setup seeds
	setup_kernel( devStates);
	do{
		// generate random numbers
		generate( devStates, rand,nbr,racN);

		if(i < k+1){
			x = rand[i];
			y= (uint)(x*x)%nbr;

			isBSmoothG(p,k,y,bsmooth);
			isInEnsembleG(div,y,sizeDiv,present);
		}
		} while(!(*bsmooth) && (*present));

		if(i<k+1){
			__syncthreads();

			if((*bsmooth) && !(*present)){
				r[i].ind.couple.x = x;
				r[i].ind.couple.y = y;

				int y1 = y;
				for(int j = 0;j<k;j++){

					while(y1%p[j] == 0){
						y1 = y1 / p[j];
						matrix[(k*i)+j]=(matrix[(k*i)+j]+1)%2;
					}
				}
				ret = 0;
			}
			__syncthreads();
		}


		free(rand);
		free(bsmooth);
		free(present);
		free(devStates);

		return ret;
	}
	__global__ void fillEnsembleG(ensemble r,int *p,int k,int nbr,int borne
			,ensemble div,int sizeDiv,int *sizeR,int *matrix){
		int res = 0;
		__shared__ int i;
		int size = 0;
		int tid=threadIdx.x;

		if( tid == 0) {
			i = 0;
			size = 0;
		}
		__syncthreads();
		res = generateRonce(r,p,k,nbr,div,sizeDiv,&size,matrix);

		if(res == 0){
			atomicAdd(&i,1);

		}
		*sizeR=i;
	}

