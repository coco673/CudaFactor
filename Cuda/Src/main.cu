#include <stdio.h>
#include <stdlib.h>
#include "header/dixon.h"

int main(int argc, char **argv) {
	Int_List_GPU *Div;
	uint64_t n;
	hipEvent_t start, stop;
	hipDeviceReset();
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	if (argc > 1)
		n = atoll(argv[1]);
	else
		n = 16;
	printf("Le nombre entré est %llu\n",n);
	if (n < 0) {
		fprintf(stderr, "Le nombre entre est négatif, entrez un nombre >= 2!\n");
		return EXIT_FAILURE;
	} else if (n <= 1) {
		printf("%llu n'est pas factorisable!\n", n);
		return EXIT_SUCCESS;
	} else if (n == 2) {
		Div = createIntList();
		addInt(&Div, n);
	} else {
		Div = dixonGPU(n);
	}
	printIntList(*Div);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("temps %f\n", elapsedTime / 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(Div);
	hipDeviceReset();
}
