#include <stdio.h>
#include <stdlib.h>
#include "header/dixon.h"

int main(int argc, char **argv) {
	Int_List_GPU *Div;
	uint64_t n;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	if (argc > 1)
		n = atoll(argv[1]);
	else
		n = 3061 * 3259;
	printf("Le nombre entré est %llu\n",n);
	if (n < 0) {
		fprintf(stderr, "Le nombre entre est négatif, entrez un nombre >= 2!\n");
		return EXIT_FAILURE;
	} else if (n <= 1) {
		printf("%llu n'est pas factorisable!\n", n);
		return EXIT_SUCCESS;
	} else if (n == 2) {
		Div = createIntList();
		addInt(&Div, n);
	} else {
		//Div = dixonGPU(n);
		Div = factor(n);
	}
	printIntList(*Div);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("temps %f\n", elapsedTime / 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	delete[](Div);
	hipDeviceReset();
}
