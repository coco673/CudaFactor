/**
 * main.cu
 */

#include <stdio.h>
#include <stdlib.h>
#include "header/dixon.h"
#include <stdint.h>

int main(int argc, char **argv) {
	Int_List_GPU *Div;
	uint64_t n;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	if (argc > 1)
	n= strtoull(argv[1],NULL,10);
	else
		n = 3061 * 3259;

	printf("Le nombre entre est %lu \n",n);

	if (n < 0) {
		fprintf(stderr, "Le nombre entre est négatif, entrez un nombre >= 2!\n");
		return EXIT_FAILURE;
	} else if (n <= 1) {
		printf("%lu n'est pas factorisable!\n", n);
		return EXIT_SUCCESS;
	} else if (n == 2) {
		Div = createIntList();
		addInt(&Div, n);
	} else {
		Div = factor(n);
	}
	printIntList(*Div);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("temps %f\n", elapsedTime / 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(Div->List);
	free(Div);
	hipDeviceReset();
}
