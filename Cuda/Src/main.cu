#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "TestFillEnsemble.h"
#include "TestPgcd.h"
#include "TestStructure.h"
#include "TestFillMatrix.h"


int main(){
	int val = 62;
	int valx = 44;
	int valy = 32;

	if(TestPgcd() == 0){
		printf("Test Pgcd -> passed\n");
	} else{
		printf(" Test Pgcd **** Not Passed\n");
	}

	if(testInitEns()== 0){
		printf("Test InitEns -> passed\n");
	} else{
		printf(" Test InitEns **** Not Passed\n");
	}
	if(testAddVal(val)== 0){
		printf("Test AddVal -> passed\n");
	} else{
		printf("Test AddVa **** Not Passed\n");
	}
	if(testAddCouple(valx,valy)== 0){
		printf("Test AddCouple -> passed\n");
	} else{
		printf("Test AddCouple **** Not Passed\n");
	}
	if(TestIsBSmooth() == 0){
		printf("Test BSmooth -> passed\n");
	} else{
		printf("Test BSmooth **** Not Passed\n");
	}
	if(TestIsInEnsemble()== 0){
		printf("Test IsInEnsemble -> passed\n");
	} else{
		printf("Test IsInEnsemble **** Not Passed\n");
	}
	if(TestIsInf()== 0){
		printf("Test isinf -> passed\n");
	} else{
		printf("Test isinf **** Not Passed\n");
	}
	if(TestIsBSmoothG()== 0){
		printf("Test BSmoothG -> passed\n");
	} else{
		printf("Test BSmoothG **** Not Passed\n");
	}
	/*if(TestIsInEnsembleG()== 0){
		printf("Test IsInEnsembleG -> passed\n");
	} else{
		printf("Test IsInEnsembleG **** Not Passed\n");
	}
	if(TestfillEnsemble() == 0){
		printf("Test fillEnsemble -> passed\n");
	} else{
		printf("Test fillEnsemble **** Not Passed\n");
	}*/
	if(TestGenerateOnce()== 0){
		printf("Test TestGenerateOnce -> passed\n");
	} else{
		printf(" Test TestGenerateOnce**** Not Passed\n");
	}
	if(TestfillEnsembleG() == 0){
		printf("Test fillEnsembleG -> passed\n");
	} else{
		printf("Test fillEnsembleG **** Not Passed\n");
	}
	if(TestFillMatrix() == 0){
			printf("Test FillMatrix -> passed\n");
		} else{
			printf("Test FillMatrix **** Not Passed\n");
		}
	return 0;
}
