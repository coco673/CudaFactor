#include "hip/hip_runtime.h"
#include "header/Dixon.h"
#include <unistd.h>
#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

int produit(ensemble div, int sizeDiv) {
	int result = 1;
	if (sizeDiv == 0) {
		return 0;
	}

	for (int i = 0; i < sizeDiv; i++) {
		result *= div[i].ind.val;
	}
	return result;
}
/*
ensemble Dixon(int n) {
	int borne = ceil(exp(sqrt(2 * log(n) * log(log(n)))));
	printf("borne = %i \n",borne);
	int cuda_result;
	int sizePrimeList;
	int *primeList = generatePrimeList(borne, &sizePrimeList);
	//sizePrimeList -= 1;
	printf("j'ai réussi a passer la fonction de merde de tim\n");
	for (int lol = 0 ; lol < sizePrimeList; lol++)printf("prime = %i",primeList[lol]);
	int sizeDiv = 0;
	int *matrix;
	int *dev_matrix;
	int *m;
	int *dev_m;
	int u, v;
	int sizeR;
	ensemble R;
	ensemble dev_R;
	ensemble div = (ensemble) malloc(sizeof(struct cell));

	//Allocation de la liste des premies sur le GPU
	int *dev_primeList;
	cuda_result = hipMalloc(&dev_primeList, sizePrimeList * sizeof(int));
	if (cuda_result != hipSuccess) {
		printf("1 %s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	//Copie de la liste des premiers
	cuda_result = hipMemcpy(dev_primeList, primeList, sizePrimeList * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_result != hipSuccess) {
		printf(" 2 %s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}

	//Allocation de l'ensemble div sur le GPU
	ensemble dev_div;
	cuda_result = hipMalloc(&dev_div, sizeof(struct cell));
	if (cuda_result != hipSuccess) {
		printf("3 %s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}

	//test primalite N
	do {
		R = (ensemble) malloc(sizePrimeList*sizeof(struct cell));

		//Allocation de l'ensemble R sur le GPU
		cuda_result = hipMalloc(&dev_R, sizePrimeList*sizeof(struct cell));
		if (cuda_result != hipSuccess) {
			printf("4 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		sizeR = 0;

		m = (int *) malloc(sizeof(int));
 *m = sizeR;
		printf("test\n");
		//Allocation de m sur le GPU
		cuda_result = hipMalloc(&dev_m, sizeof(int));
		if (cuda_result != hipSuccess) {
			printf("5 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		//Copie de m sur le GPU
		cuda_result = hipMemcpy(dev_m, m, sizeof(int), hipMemcpyHostToDevice);
		if (cuda_result != hipSuccess) {
			printf(" 6 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		matrix = (int *) malloc(sizePrimeList * sizePrimeList * sizeof(int));


		//Allocation de la matrice sur le GPU
		cuda_result = hipMalloc(&dev_matrix, sizePrimeList * sizePrimeList * sizeof(int));
		if (cuda_result != hipSuccess) {
			printf("7 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		cuda_result = hipMemcpy(dev_div, div, sizeDiv * sizeof(int), hipMemcpyHostToDevice);
		if (cuda_result != hipSuccess) {
			printf(" 8 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		printf("CUDA? ES TU LA ?\n");
		fillEnsembleG<<<1, sizePrimeList>>>(dev_R, dev_primeList, sizePrimeList, n, borne, dev_div, sizeDiv, dev_m, dev_matrix);
		printf("NON\n");
		//Recopie de la matrice resultante
		cuda_result = hipMemcpy(matrix, dev_matrix, sizePrimeList * sizePrimeList * sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("9 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de m
		cuda_result = hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("10 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de l'ensemble R
		cuda_result = hipMemcpy(R, dev_R, *m * sizeof(struct cell), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("11 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		printf("OUUUUUUI !\n");
		//Recopie de la liste des premiers
		cuda_result = hipMemcpy(primeList, dev_primeList, sizePrimeList * sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf(" 12 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		printf("GAUSS ??\n");
		int *E = gaussjordan_noyau(matrix, sizePrimeList);
		u = initU(R, *m, E, n);
		v = calcul_v(primeList, sizePrimeList, *m, matrix, sizePrimeList, E);
		if (pgcdUint(u - v, n) != 1 && pgcdUint(u - v, n) != n) {
			addVal(&div, u - v, &sizeDiv);
		} else if (pgcdUint(u + v, n) != 1 && pgcdUint(u + v, n) != n) {
			addVal(&div, u + v, &sizeDiv);
		}
		printf("CALL ME MAYBE ??\n");
		free(R);
		printf("BOUM R\n");
		hipFree(dev_R);
		printf("BOUM DEV_R\n");
		free(m);
		printf("BOUM M\n");
		hipFree(dev_m);
		printf("BOUM DEVM\n");
		free(matrix);
		printf("PILLULE ROUGE\n");
		hipFree(dev_matrix);
		printf("PILLULE BLEUE ! sizeDiv = %i  \n",sizeDiv);
	}while (produit(div, sizeDiv) != n);
	printf("HEY\n");
	for(int i = 0; i<sizeDiv;i++){
		printf("res = %i\n",div[i].ind.val);

	}
	return div;
}
 */
ensemble Dixon2(int n,int *sizefinal){
	int borne = ceil(exp(sqrt(2 * log(n) * log(log(n)))));
	int nbr = n;
	int sizePrimeList;
	int *primeList = generatePrimeList(borne, &sizePrimeList);
	printf("sizePrimeList = %i\n",sizePrimeList);
	int sqrtNRB = (int)sqrt(nbr);
	int *rand = (int *) malloc(sizePrimeList*sizeof(int));
	int *dev_rand;
	hiprandState_t *state = (hiprandState_t *)malloc(sizePrimeList*sizeof(hiprandState_t));
	hiprandState_t *dev_state ;
	int *dev_primeList;
	CUDA_CHECK_RETURN(hipMalloc((int **)&dev_primeList, sizePrimeList * sizeof(int)));
	CUDA_CHECK_RETURN(hipMemcpy(dev_primeList, primeList, sizePrimeList * sizeof(int), hipMemcpyHostToDevice));
	ensemble R, dev_R;
	ensemble Div = (ensemble) malloc(sizeof(struct cell));
	int *sizeDiv = (int*)malloc(sizeof(int));
	*sizeDiv = 0;
	ensemble dev_Div;
	int *sizeR=(int*)malloc(sizeof(int));
	int *dev_sizeR;
	int *dev_sizeDiv;
	printf("bonjour\n");
	for(int j = 0; j < sizePrimeList; j++){
		while(nbr % primeList[j] == 0){
			nbr = (nbr / primeList[j]);
			addVal(&Div, primeList[j], sizeDiv);
			printf("ok  size div =%i nbr = %i\n",*sizeDiv,nbr);
		}
	}
	if (nbr == 1){
		*sizefinal = *sizeDiv;
		return Div;
	}
	int *matrix;
	int *dev_matrix;
	int *kernel;
	int u, v;
	u = v = 1;
	printf("la borne est %i\n",borne);
	while(produit(Div,*sizeDiv) != n){
		printf("malloc 1\n");
		matrix= (int*)malloc(sizePrimeList * sizePrimeList * sizeof(int));
		printf("malloc 2\n");
		R = (ensemble) malloc(sizePrimeList * sizeof(struct cell));
		*sizeR = 0;
		CUDA_CHECK_RETURN(hipMalloc(&dev_rand,sizePrimeList*sizeof(int)));
		CUDA_CHECK_RETURN(hipMemcpy(dev_rand, rand, sizePrimeList * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMalloc((int **)&dev_sizeR, sizePrimeList*sizeof(int)));
		CUDA_CHECK_RETURN(hipMalloc((int **)&dev_sizeDiv, sizeof(int)));
		CUDA_CHECK_RETURN(hipMalloc((hiprandState_t**)&dev_state, sizePrimeList*sizeof(hiprandState_t)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR, 0, sizeof(int)));
		CUDA_CHECK_RETURN(hipMalloc((ensemble *)&dev_R, sizePrimeList * sizeof(struct cell)));
		CUDA_CHECK_RETURN(hipMalloc((ensemble *)&dev_Div, *sizeDiv * sizeof(struct cell)));
		CUDA_CHECK_RETURN(hipMemcpy(dev_R, R, *sizeR * sizeof(struct cell), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(dev_sizeR, sizeR,sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(dev_sizeDiv, sizeDiv,sizeof(int), hipMemcpyHostToDevice));

		printf("size Div %i\n",*sizeDiv);
		printf(" Div %i\n",Div[*sizeDiv-1].ind.val);
		CUDA_CHECK_RETURN(hipMemcpy(dev_Div, Div, *sizeDiv * sizeof(struct cell), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMalloc((int **)&dev_matrix, sizePrimeList * sizePrimeList * sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix, 0, sizePrimeList * sizePrimeList * sizeof(int)));
printf("on ouvre %i thread \n",sizePrimeList);
		Generation<<<1,sizePrimeList>>>(dev_state,nbr, sqrtNRB,dev_rand);
		printf("size Div = %i\n",*sizeDiv);
		printf("on ouvre %i thread pour remplir \n",sizePrimeList);
		fillEnsR<<<1,sizePrimeList>>>(dev_state,dev_R,dev_sizeR,dev_Div,dev_sizeDiv,dev_primeList,sizePrimeList,dev_rand,nbr,dev_matrix);
		printf("OK T ES LA \n");

		CUDA_CHECK_RETURN(hipMemcpy(sizeR, dev_sizeR, sizeof(int), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(sizeDiv, dev_sizeR, sizeof(int), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(R, dev_R, *sizeR * sizeof(struct cell), hipMemcpyDeviceToHost));
printf("ta frangine !! \n");
		CUDA_CHECK_RETURN(hipMemcpy(matrix, dev_matrix, sizePrimeList * sizePrimeList * sizeof(int), hipMemcpyDeviceToHost));
		int *kernel = (int *) malloc(sizePrimeList * sizeof(int));

		gaussjordan_noyau(matrix, sizePrimeList,kernel);
		int tmp = 1;
		printf("malloc 3 + size of R = %i\n",*sizeR);

		//int *test = (int*) malloc((*sizeR)*sizeof(int));
		int test[N];
		printf("memset 1\n");

		memset(test,0,(*sizeR)*sizeof(int));
		printf("for 1\n");

		for(int i = 0; i < *sizeR; i++){
			if(kernel[i] != 0){
				tmp = tmp * (R[i].ind.couple.y);
				v = v * (R[i].ind.couple.x);
			}
		}
		printf("FIN U\n");
		u = 1;
		for(int i = 0; i < *sizeR; i++){
			while(tmp % primeList[i]){
				tmp = (tmp / primeList[i]);
				test[i] += 1;
			}
			test[i] = floor(test[i] / 2);
			u = u * (primeList[i] * test[i]);
		}
		printf("FIN V\n");
		if (pgcdUint(u+v,nbr) != 1){
			addVal(&Div, (u+v), sizeDiv);
		}
		printf("FIN PGCD\n");
		//CUDA_CHECK_RETURN(hipFree(dev_Div));
		printf("free -1\n");

		//CUDA_CHECK_RETURN(hipFree(dev_sizeDiv));
		printf("free -21\n");

		//CUDA_CHECK_RETURN(hipFree(dev_R));
		printf("free -31\n");

		//CUDA_CHECK_RETURN(hipFree(dev_matrix));
		printf("free -41\n");

		//CUDA_CHECK_RETURN(hipFree(dev_sizeR));
		printf("free 1\n");
		free(R);
		printf("free 2\n");
		//free(test);
		printf("free 3\n");
		//free(matrix);
		printf("free 4\n");
		//free(kernel);
		printf("free 5\n");
	}
	*sizefinal = *sizeDiv;
	return Div;
}

ensemble Dixon(int n) {
	int borne = ceil(exp(sqrt(2 * log(n) * log(log(n)))));
	int sizePremList;
	int *premList = generatePrimeList(borne, &sizePremList);
	int *dev_premList;
	CUDA_CHECK_RETURN(hipMalloc((int **)&dev_premList, sizePremList * sizeof(int)));
	CUDA_CHECK_RETURN(hipMemcpy(dev_premList, premList, sizePremList * sizeof(int), hipMemcpyHostToDevice));
	ensemble R, dev_R;
	ensemble Div = (ensemble) malloc(sizeof(struct cell));
	int sizeDiv = 0;
	ensemble dev_Div;
	int sizeR;
	int *dev_sizeR;
	int *matrix;
	int *dev_matrix;
	int *kernel;
	int u, v;
	//test de primalite
	while (produit(Div, sizeDiv) != n) {
		matrix = (int *) malloc(sizePremList * sizePremList * sizeof(int));
		R = (ensemble) malloc(sizePremList * sizeof(struct cell));
		sizeR = 0;

		CUDA_CHECK_RETURN(hipMalloc((int **)&dev_sizeR, sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_sizeR, 0, sizeof(int)));
		CUDA_CHECK_RETURN(hipMalloc((ensemble *)&dev_R, sizeR * sizeof(struct cell)));
		CUDA_CHECK_RETURN(hipMemcpy(dev_R, R, sizeR * sizeof(struct cell), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(dev_Div, Div, sizeDiv * sizeof(struct cell), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMalloc((int **)&dev_matrix, sizePremList * sizePremList * sizeof(int)));
		CUDA_CHECK_RETURN(hipMemset(dev_matrix, 0, sizePremList * sizePremList * sizeof(int)));

		//fillEnsembleG<<<1, sizePremList>>>(dev_R, dev_premList, sizePremList, n, borne, Div, sizeDiv, dev_sizeR, dev_matrix);

		CUDA_CHECK_RETURN(hipMemcpy(matrix, dev_matrix, sizePremList * sizePremList * sizeof(int), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&sizeR, dev_sizeR, sizeof(int), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(R, dev_R, sizeR * sizeof(struct cell), hipMemcpyDeviceToHost));
		//kernel = gaussjordan_noyau(matrix, sizePremList);
		u = initU(R, sizeR, kernel, n);
		v = calcul_v(premList, sizePremList, sizeR, matrix, sizePremList, kernel);
		if (pgcdUint(u - v, n) != 1 && pgcdUint(u - v, n) != n) {
			addVal(&Div, u - v, &sizeDiv);
		} else if (pgcdUint(u + v, n) != 1 && pgcdUint(u + v, n) != n) {
			addVal(&Div, u + v, &sizeDiv);
		}
		free(R);
		CUDA_CHECK_RETURN(hipFree(dev_R));
		CUDA_CHECK_RETURN(hipFree(dev_sizeR));
		free(matrix);
		CUDA_CHECK_RETURN(hipFree(dev_matrix));
		printf("Passage");
	}
	return Div;
}
