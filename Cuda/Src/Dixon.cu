#include "hip/hip_runtime.h"
#include "header/Dixon.h"


int produit(ensemble div, int sizeDiv) {
	int result = 1;
	if (sizeDiv == 0) {
		return 0;
	}

	for (int i = 0; i < sizeDiv; i++) {
		result *= div[i].ind.val;
	}
	return result;
}

ensemble Dixon(int n) {
	int borne = ceil(exp(sqrt(2 * log(n) * log(log(n)))));
	printf("borne = %i \n",borne);
	int cuda_result;
	int sizePrimeList;
	int *primeList = generatePrimeList(borne, &sizePrimeList);
	//sizePrimeList -= 1;
	printf("j'ai réussi a passer la fonction de merde de tim\n");
	for (int lol = 0 ; lol < sizePrimeList; lol++)printf("prime = %i",primeList[lol]);
	int sizeDiv = 0;
	int *matrix;
	int *dev_matrix;
	int *m;
	int *dev_m;
	int u, v;
	int sizeR;
	ensemble R;
	ensemble dev_R;
	ensemble div = (ensemble) malloc(sizeof(struct cell));

	//Allocation de la liste des premies sur le GPU
	int *dev_primeList;
	cuda_result = hipMalloc(&dev_primeList, sizePrimeList * sizeof(int));
	if (cuda_result != hipSuccess) {
		printf("1 %s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	//Copie de la liste des premiers
	cuda_result = hipMemcpy(dev_primeList, primeList, sizePrimeList * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_result != hipSuccess) {
		printf(" 2 %s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}

	//Allocation de l'ensemble div sur le GPU
	ensemble dev_div;
	cuda_result = hipMalloc(&dev_div, sizeof(struct cell));
	if (cuda_result != hipSuccess) {
		printf("3 %s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}

	//test primalite N
	do {
		R = (ensemble) malloc(sizePrimeList*sizeof(struct cell));

		//Allocation de l'ensemble R sur le GPU
		cuda_result = hipMalloc(&dev_R, sizePrimeList*sizeof(struct cell));
		if (cuda_result != hipSuccess) {
			printf("4 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		sizeR = 0;

		m = (int *) malloc(sizeof(int));
		*m = sizeR;
		printf("test\n");
		//Allocation de m sur le GPU
		cuda_result = hipMalloc(&dev_m, sizeof(int));
		if (cuda_result != hipSuccess) {
			printf("5 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		//Copie de m sur le GPU
		cuda_result = hipMemcpy(dev_m, m, sizeof(int), hipMemcpyHostToDevice);
		if (cuda_result != hipSuccess) {
			printf(" 6 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		matrix = (int *) malloc(sizePrimeList * sizePrimeList * sizeof(int));


		//Allocation de la matrice sur le GPU
		cuda_result = hipMalloc(&dev_matrix, sizePrimeList * sizePrimeList * sizeof(int));
		if (cuda_result != hipSuccess) {
			printf("7 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		cuda_result = hipMemcpy(dev_div, div, sizeDiv * sizeof(int), hipMemcpyHostToDevice);
		if (cuda_result != hipSuccess) {
			printf(" 8 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		printf("CUDA? ES TU LA ?\n");
		fillEnsembleG<<<1, sizePrimeList>>>(dev_R, dev_primeList, sizePrimeList, n, borne, dev_div, sizeDiv, dev_m, dev_matrix);
		printf("NON\n");
		//Recopie de la matrice resultante
		cuda_result = hipMemcpy(matrix, dev_matrix, sizePrimeList * sizePrimeList * sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("9 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de m
		cuda_result = hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("10 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de l'ensemble R
		cuda_result = hipMemcpy(R, dev_R, *m * sizeof(struct cell), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("11 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		printf("OUUUUUUI !\n");
		//Recopie de la liste des premiers
		cuda_result = hipMemcpy(primeList, dev_primeList, sizePrimeList * sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf(" 12 %s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		printf("GAUSS ??\n");
		int *E = gaussjordan_noyau(matrix, sizePrimeList);
		u = initU(R, *m, E, n);
		v = calcul_v(primeList, sizePrimeList, *m, matrix, sizePrimeList, E);
		if (pgcdUint(u - v, n) != 1 && pgcdUint(u - v, n) != n) {
			addVal(&div, u - v, &sizeDiv);
		} else if (pgcdUint(u + v, n) != 1 && pgcdUint(u + v, n) != n) {
			addVal(&div, u + v, &sizeDiv);
		}
		printf("CALL ME MAYBE ??\n");
		free(R);
		printf("BOUM R\n");
		hipFree(dev_R);
		printf("BOUM DEV_R\n");
		free(m);
		printf("BOUM M\n");
		hipFree(dev_m);
		printf("BOUM DEVM\n");
		free(matrix);
		printf("PILLULE ROUGE\n");
		hipFree(dev_matrix);
		printf("PILLULE BLEUE ! sizeDiv = %i  \n",sizeDiv);
	}while (produit(div, sizeDiv) != n);
	printf("HEY\n");
	for(int i = 0; i<sizeDiv;i++){
		printf("res = %i\n",div[i].ind.val);

	}
	return div;
}
