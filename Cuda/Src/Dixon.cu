#include "hip/hip_runtime.h"
#include "header/Dixon.h"


int produit(ensemble div, int sizeDiv) {
	int result = 1;
	if (sizeDiv == 0) {
		return 0;
	}
	for (int i = 0; i < sizeDiv; i++) {
		result *= div[i].ind.val;
	}
	return result;
}

ensemble Dixon(int n) {
	int borne = ceil(exp(sqrt(2 * log(n) * log(log(n)))));
	int cuda_result;
	int sizePrimeList;
	int *primeList = generatePrimeList(borne, &sizePrimeList);
	int sizeDiv = 0;
	int *matrix;
	int *dev_matrix;
	int *m;
	int *dev_m;
	int u, v;
	int sizeR;
	ensemble R;
	ensemble dev_R;
	ensemble div = (ensemble) malloc(sizeof(struct cell));

	//Allocation de la liste des premies sur le GPU
	int *dev_primeList;
	cuda_result = hipMalloc(&dev_primeList, sizePrimeList * sizeof(int));
	if (cuda_result != hipSuccess) {
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}
	//Copie de la liste des premiers
	cuda_result = hipMemcpy(dev_primeList, primeList, sizePrimeList * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_result != hipSuccess) {
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}

	//Allocation de l'ensemble div sur le GPU
	ensemble dev_div;
	cuda_result = hipMalloc(&dev_div, sizeof(struct cell));
	if (cuda_result != hipSuccess) {
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		return NULL;
	}

	//test primalite N
	while (produit(div, sizeDiv) != n) {
		R = (ensemble) malloc(sizeof(struct cell));

		//Allocation de l'ensemble R sur le GPU
		cuda_result = hipMalloc(&dev_R, sizeof(struct cell));
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		sizeR = 0;

		m = (int *) malloc(sizeof(int));
		*m = sizeR;

		//Allocation de m sur le GPU
		cuda_result = hipMalloc(&dev_m, sizeof(int));
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}
		//Copie de m sur le GPU
		cuda_result = hipMemcpy(dev_m, m, sizeof(int), hipMemcpyHostToDevice);
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		matrix = (int *) malloc(sizePrimeList * sizePrimeList * sizeof(int));


		//Allocation de la matrice sur le GPU
		cuda_result = hipMalloc(&dev_matrix, sizePrimeList * sizePrimeList * sizeof(int));
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		cuda_result = hipMemcpy(dev_div, div, sizeDiv * sizeof(int), hipMemcpyHostToDevice);
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		fillEnsembleG<<<1, sizePrimeList>>>(dev_R, dev_primeList, sizePrimeList, n, borne, dev_div, sizeDiv, m, dev_matrix);

		//Recopie de la matrice resultante
		cuda_result = hipMemcpy(matrix, dev_matrix, sizePrimeList * sizePrimeList * sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de m
		cuda_result = hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de l'ensemble R
		cuda_result = hipMemcpy(R, dev_R, *m * sizeof(struct cell), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		//Recopie de la liste des premiers
		cuda_result = hipMemcpy(primeList, dev_primeList, sizePrimeList * sizeof(int), hipMemcpyDeviceToHost);
		if (cuda_result != hipSuccess) {
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			return NULL;
		}

		int *E = gaussjordan_noyau(matrix, sizePrimeList);
		u = initU(R, *m, E, n);
		v = calcul_v(primeList, sizePrimeList, *m, matrix, sizePrimeList, E);
		if (pgcdUint(u - v, n) != 1 && pgcdUint(u - v, n) != n) {
			addVal(&div, u - v, &sizeDiv);
		} else if (pgcdUint(u + v, n) != 1 && pgcdUint(u + v, n) != n) {
			addVal(&div, u + v, &sizeDiv);
		}
		free(R);
		hipFree(dev_R);
		free(m);
		hipFree(dev_m);
		free(matrix);
		hipFree(dev_matrix);
	}
	return div;
}
