#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


int calcul_v(int *premList, int sizePremList, int m, int *matrix, int sizeMatrix, int *noyau) {
	int res_somme;
	int res_produit = 1;
	for (int i = 0; i < sizePremList; i++) {
		res_somme = 0;
		for (int j = 0; j < m; j++) {
			res_somme += matrix[j * sizeMatrix + i] * noyau[j];
		}
		res_somme /= 2;
		res_produit *= pow(premList[i], res_somme);
	}
	return res_produit;
}
