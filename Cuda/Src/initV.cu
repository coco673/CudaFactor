#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "initV.h"

int calcul_v(uint64_t *premList, int sizePremList, int m, int *matrix, int sizeMatrix, int *noyau) {
	int res_somme;
	int res_produit = 1;
	for (int i = 0; i < sizePremList; i++) {
		res_somme = 0;
		for (int j = 0; j < m; j++) {
			res_somme += matrix[j * sizeMatrix + i] * noyau[j];
		}
		res_somme /= 2;
		res_produit *= pow(premList[i], res_somme);
	}
	return res_produit;
}
