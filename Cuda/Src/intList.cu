#include "hip/hip_runtime.h"
#include "intList.h"

/*
__device__ __host__ Int_List *createIntList() {
	Int_List *list = (Int_List *) malloc(sizeof(Int_List));
	list->list = NULL;
	list->size = 0;
	return list;
}

__device__ __host__ void addInt(Int_List *list, int c) {
	if (list->size == 0) {
		list->list = (struct IL*) malloc(sizeof(struct IL));
		list->list->val = c;
		list->list->suiv = NULL;
	} else {
		struct IL *cl = (struct IL*) malloc(sizeof(struct IL));
		cl->val = c;
		cl->suiv = list->list;
		list->list = cl;
	}
	list->size++;
}

__device__ __host__ int getVal(const Int_List list, int index) {
	int i = 0;
	Int_List tmp = list;
	if (index > list.size - 1 || index < 0) {
		return -1;
	} else {
		while (i != index) {
			tmp.list = tmp.list->suiv;
			i++;
		}
		return tmp.list->val;
	}
}

__device__ __host__ void removeLastInt(Int_List *list) {
	if (list->size != 0) {
		struct IL *ptr;
		ptr = list->list;
		while (ptr->suiv != NULL) {
			if (ptr->suiv->suiv == NULL) {
				struct IL *tmp;
				tmp = ptr;
				ptr = ptr->suiv;
				tmp->suiv = NULL;
			} else {
				ptr = ptr->suiv;
			}
		}
		free(ptr);
		list->size--;
	}
}

__device__ __host__ void resetIntList(Int_List *list) {
	while (list->size != 0) {
		removeLastInt(list);
	}
}

void printIntList(const Int_List list) {
	printf("Taille de la liste : %i\n", list.size);
	for (int i = 0; i < list.size; i++) {
		printf("val : %i\n", getVal(list, i));
	}
}
*/

__device__ void copyTabDev(int *src, int *dest, int size) {
	if (blockIdx.x == 0) {
		int tid = threadIdx.x;
		if (tid < size) {
			dest[tid] = src[tid];
		}
	}
}

//1 block ; size thread
__global__ void copyTabGPU(int *src, int *dest, int size) {
	int tid = threadIdx.x;
	if (tid < size) {
		dest[tid] = src[tid];
	}
}

__host__ Int_List_GPU *createIntList() {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->Size = 0;
	l->List = NULL;
	return l;
}

__host__ void addInt(Int_List_GPU **list, int v) {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->List = new int[(*list)->Size + 1];
	l->Size = (*list)->Size + 1;
	int *dev_list_dest, *dev_list_src;
	hipMalloc((void **)&dev_list_src, (*list)->Size * sizeof(int));
	hipMemcpy(dev_list_src, (*list)->List, (*list)->Size * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_list_dest, ((*list)->Size + 1) * sizeof(int));
	copyTabGPU<<<1, (*list)->Size>>>(dev_list_src, dev_list_dest, (*list)->Size);
	hipMemcpy(l->List, dev_list_dest, (*list)->Size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_list_src);
	hipFree(dev_list_dest);
	l->List[(*list)->Size] = v;

	delete((*list)->List);
	delete(*list);
	*list = l;
}

__device__ void addIntGPU(int **list, int size, int v) {
	if (blockIdx.x == 0) {
		__shared__ int *l;
		if (threadIdx.x == 0) {
			l = new int[size + 1];
		}
		copyTabDev((*list), l, size);
		l[size] = v;
		//delete(list);
		*list = l;
	}
}

__host__ int getVal(Int_List_GPU l, int index) {
	return l.List[index];
}

__device__ int getValGPU(int *l, int index) {
	return l[index];
}

__host__ void removeLastInt(Int_List_GPU **list) {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->List = new int[(*list)->Size - 1];
	int *dev_list_dest, *dev_list_src;
	hipMalloc((void **)&dev_list_src, (*list)->Size * sizeof(int));
	hipMemcpy(dev_list_src, (*list)->List, (*list)->Size * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_list_dest, ((*list)->Size - 1) * sizeof(int));
	copyTabGPU<<<1, (*list)->Size - 1>>>((*list)->List, l->List, (*list)->Size - 1);
	hipMemcpy(l->List, dev_list_dest, ((*list)->Size - 1) * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_list_src);
	hipFree(dev_list_dest);
	l->Size = (*list)->Size - 1;
	//delete(list->List);
	//delete(list);
	*list = l;
}

__device__ void removeLastInt(int **list, int size) {
	if (blockIdx.x == 0) {
		__shared__ int *l;
		if (threadIdx.x == 0) {
			l = new int[size - 1];
		}
		copyTabDev((*list), l, size - 1);
		//delete(list);
		*list = l;
	}
}

__host__ void resetIntList(Int_List_GPU **list) {
	while ((*list)->Size > 0) {
		removeLastInt(list);
	}
}

__device__ void resetIntListGPU(int **list, int size) {
	if (blockIdx.x == 0) {
		for (int i = 0; i < size; i++) {
			removeLastInt(list, size);
		}
	}
}

__host__ void printIntList(Int_List_GPU l) {
	printf("Taille de la liste : %i\n", l.Size);
	for (int i = 0; i < l.Size; i++) {
		printf("valeur : %i\n", getVal(l, i));
	}
}
