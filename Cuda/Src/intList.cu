#include "hip/hip_runtime.h"
#include "header/intList.h"

/*
__device__ __host__ Int_List *createIntList() {
	Int_List *list = (Int_List *) malloc(sizeof(Int_List));
	list->list = NULL;
	list->size = 0;
	return list;
}

__device__ __host__ void addInt(Int_List *list, int c) {
	if (list->size == 0) {
		list->list = (struct IL*) malloc(sizeof(struct IL));
		list->list->val = c;
		list->list->suiv = NULL;
	} else {
		struct IL *cl = (struct IL*) malloc(sizeof(struct IL));
		cl->val = c;
		cl->suiv = list->list;
		list->list = cl;
	}
	list->size++;
}

__device__ __host__ int getVal(const Int_List list, int index) {
	int i = 0;
	Int_List tmp = list;
	if (index > list.size - 1 || index < 0) {
		return -1;
	} else {
		while (i != index) {
			tmp.list = tmp.list->suiv;
			i++;
		}
		return tmp.list->val;
	}
}

__device__ __host__ void removeLastInt(Int_List *list) {
	if (list->size != 0) {
		struct IL *ptr;
		ptr = list->list;
		while (ptr->suiv != NULL) {
			if (ptr->suiv->suiv == NULL) {
				struct IL *tmp;
				tmp = ptr;
				ptr = ptr->suiv;
				tmp->suiv = NULL;
			} else {
				ptr = ptr->suiv;
			}
		}
		free(ptr);
		list->size--;
	}
}

__device__ __host__ void resetIntList(Int_List *list) {
	while (list->size != 0) {
		removeLastInt(list);
	}
}

void printIntList(const Int_List list) {
	printf("Taille de la liste : %i\n", list.size);
	for (int i = 0; i < list.size; i++) {
		printf("val : %i\n", getVal(list, i));
	}
}
*/

__device__ void copyTabDev(uint64_t *src, uint64_t *dest, int size) {
	if (blockIdx.x == 0) {
		int tid = threadIdx.x;
		if (tid < size) {
			dest[tid] = src[tid];
		}
	}
}

//1 block ; size thread
__global__ void copyTabGPU(uint64_t *src, uint64_t *dest, int size) {
	int tid = threadIdx.x;
	if (tid < size) {
		dest[tid] = src[tid];
	}
}

__host__ Int_List_GPU *createIntList() {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->Size = 0;
	l->List = NULL;
	return l;
}

__host__ void addInt(Int_List_GPU **list, int v) {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->List = new uint64_t[(*list)->Size + 1];
	l->Size = (*list)->Size + 1;
	uint64_t *dev_list_dest, *dev_list_src;
	hipMalloc((void **)&dev_list_src, (*list)->Size * sizeof(uint64_t));
	hipMemcpy(dev_list_src, (*list)->List, (*list)->Size * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_list_dest, ((*list)->Size + 1) * sizeof(uint64_t));
	copyTabGPU<<<1, (*list)->Size>>>(dev_list_src, dev_list_dest, (*list)->Size);
	hipMemcpy(l->List, dev_list_dest, (*list)->Size * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipFree(dev_list_src);
	hipFree(dev_list_dest);
	l->List[(*list)->Size] = v;

	delete[]((list[0])->List);
	delete[](list[0]);
	list[0] = l;
}

__device__ void addIntGPU(uint64_t **list, int size, int v) {
	if (blockIdx.x == 0) {
		__shared__ uint64_t *l;
		if (threadIdx.x == 0) {
			l = new uint64_t[size + 1];
		}
		copyTabDev((*list), l, size);
		l[size] = v;
		//delete(list);
		*list = l;
	}
}

__host__ uint64_t getVal(Int_List_GPU l, int index) {
	return l.List[index];
}

__device__ uint64_t getValGPU(uint64_t *l, int index) {
	return l[index];
}

__host__ void removeLastInt(Int_List_GPU **list) {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->List = new uint64_t[(*list)->Size - 1];
	int *dev_list_dest, *dev_list_src;
	hipMalloc((void **)&dev_list_src, (*list)->Size * sizeof(int));
	hipMemcpy(dev_list_src, (*list)->List, (*list)->Size * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_list_dest, ((*list)->Size - 1) * sizeof(int));
	copyTabGPU<<<1, (*list)->Size - 1>>>((*list)->List, l->List, (*list)->Size - 1);
	hipMemcpy(l->List, dev_list_dest, ((*list)->Size - 1) * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_list_src);
	hipFree(dev_list_dest);
	l->Size = (*list)->Size - 1;
	//delete(list->List);
	//delete(list);
	*list = l;
}

__device__ void removeLastInt(uint64_t **list, uint64_t size) {
	if (blockIdx.x == 0) {
		__shared__ uint64_t *l;
		if (threadIdx.x == 0) {
			l = new uint64_t[size - 1];
		}
		copyTabDev((*list), l, size - 1);
		//delete(list);
		*list = l;
	}
}

__host__ void resetIntList(Int_List_GPU **list) {
	while ((*list)->Size > 0) {
		removeLastInt(list);
	}
}

__device__ void resetIntListGPU(uint64_t **list, uint64_t size) {
	if (blockIdx.x == 0) {
		for (int i = 0; i < size; i++) {
			removeLastInt(list, size);
		}
	}
}

__host__ void printIntList(Int_List_GPU l) {
	printf("%i\n", l.Size);
	char* tmp = (char *) malloc (1000*sizeof(char));
	char *tmptmp = (char *) malloc (1000*sizeof(char));
	sprintf(tmp,"%s ","facteurs");
	for (int i = 0; i < l.Size; i++) {
		//printf("valeur : %ld\n", getVal(l, i));
		sprintf(tmptmp," %llu", getVal(l, i));
		strcat(tmp, tmptmp);
		sprintf(tmptmp,"%s", "");
	}
	printf("%s\n",tmp);
	free(tmptmp);
	free(tmp);
}
