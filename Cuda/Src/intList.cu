#include "hip/hip_runtime.h"
/**
 * intList.cu
 */
#include "header/intList.h"

__device__ void copyTabDev(uint64_t *src, uint64_t *dest, int size) {
	if (blockIdx.x == 0) {
		int tid = threadIdx.x;
		if (tid < size) {
			dest[tid] = src[tid];
		}
	}
}

//1 block ; size thread
__global__ void copyTabGPU(uint64_t *src, uint64_t *dest, int size) {
	int tid = threadIdx.x;
	if (tid < size) {
		 dest[tid] = src[tid];
	}
}

__host__ Int_List_GPU *createIntList() {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->Size = 0;
	l->List = NULL;
	return l;
}

__host__ void addInt(Int_List_GPU **list, int v) {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->List = new uint64_t[(*list)->Size + 1];
	l->Size = (*list)->Size + 1;
	uint64_t *dev_list_dest, *dev_list_src;

	hipMalloc((void **)&dev_list_src, (*list)->Size * sizeof(uint64_t));
	hipMemcpy(dev_list_src, (*list)->List, (*list)->Size * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_list_dest, ((*list)->Size + 1) * sizeof(uint64_t));
	
	copyTabGPU<<<1, (*list)->Size>>>(dev_list_src, dev_list_dest, (*list)->Size);
	
	hipMemcpy(l->List, dev_list_dest, (*list)->Size * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipFree(dev_list_src);
	hipFree(dev_list_dest);
	
	l->List[(*list)->Size] = v;

	delete[]((list[0])->List);
	delete[](list[0]);
	list[0] = l;
}

__device__ void addIntGPU(uint64_t **list, int size, int v) {
	if (blockIdx.x == 0) {
		__shared__ uint64_t *l;
		if (threadIdx.x == 0) {
			l = new uint64_t[size + 1];
		}
		copyTabDev((*list), l, size);
		l[size] = v;
		*list = l;
	}
}

__host__ uint64_t getVal(Int_List_GPU l, int index) {
	return l.List[index];
}

__device__ uint64_t getValGPU(uint64_t *l, int index) {
	return l[index];
}

__host__ void removeLastInt(Int_List_GPU **list) {
	Int_List_GPU *l = new Int_List_GPU[1];
	l->List = new uint64_t[(*list)->Size - 1];
	int *dev_list_dest, *dev_list_src;
	
	hipMalloc((void **)&dev_list_src, (*list)->Size * sizeof(int));
	hipMemcpy(dev_list_src, (*list)->List, (*list)->Size * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_list_dest, ((*list)->Size - 1) * sizeof(int));
	
	copyTabGPU<<<1, (*list)->Size - 1>>>((*list)->List, l->List, (*list)->Size - 1);
	
	hipMemcpy(l->List, dev_list_dest, ((*list)->Size - 1) * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_list_src);
	hipFree(dev_list_dest);
	
	l->Size = (*list)->Size - 1;
	*list = l;
}

__device__ void removeLastInt(uint64_t **list, uint64_t size) {
	if (blockIdx.x == 0) {
		__shared__ uint64_t *l;
		if (threadIdx.x == 0) {
			l = new uint64_t[size - 1];
		}
		copyTabDev((*list), l, size - 1);
		*list = l;
	}
}

__host__ void resetIntList(Int_List_GPU **list) {
	while ((*list)->Size > 0) {
		removeLastInt(list);
	}
}

__device__ void resetIntListGPU(uint64_t **list, uint64_t size) {
	if (blockIdx.x == 0) {
		for (int i = 0; i < size; i++) {
			removeLastInt(list, size);
		}
	}
}

__host__ void printIntList(Int_List_GPU l) {
	printf("%i\n", l.Size);
	char* tmp = (char *) malloc (1000*sizeof(char));
	char *tmptmp = (char *) malloc (1000*sizeof(char));
	sprintf(tmp,"%s ","facteurs");
	for (int i = 0; i < l.Size; i++) {
		sprintf(tmptmp,"%lu ", getVal(l, i));
		strcat(tmp, tmptmp);
		sprintf(tmptmp,"%s", "");
	}
	printf("%s\n",tmp);
	free(tmptmp);
	free(tmp);
}

uint64_t produitDiv(Int_List_GPU Div) {
	uint64_t res = 1;
	for (int i = 0; i < Div.Size; i++) {
		res *= getVal(Div, i);
	}
	return res;
}

int notIn(Int_List_GPU Div, uint64_t val) {
	for (int i = 0; i < Div.Size; i++) {
		if (getVal(Div, i) == val) {
			return 1;
		}
	}
	return 0;
}

Int_List_GPU *mergeDiv(Int_List_GPU *src1, Int_List_GPU *src2) {
	Int_List_GPU *result = createIntList();
	for (int i = 0; i < src1->Size; i++) {
		addInt(&result, src1->List[i]);
	}
	for (int i = 0; i < src2->Size; i++) {
		addInt(&result, src2->List[i]);
	}
	free(src1);
	free(src2);
	return result;
}
