#include "hip/hip_runtime.h"
/*
 * init.cu
 *
 *  Created on: 27 mars 2014
 *      Author: clement
 */



#include "header/initU.h"

/*
__global__ void prodPuissG(int *x, int puiss, int *res, int block) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < puiss-1) {
		res[id] = x[block] * x[block];
	} else {
		res[id] = x[block];
	}
}

__global__ void sommeG(int *e, int *som, int size) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
		if (id < size-1) {
			som[id] = e[2*id] + e[2*id+1];
		} else {
			som[id] = e[2*id];
		}
}*/

int initU(int *x, int m, int *e) {
	int res = 1;
	for (int i = 0; i < m; i++) {
		if (e[i] != 0) {
			res = res * (x[i] * x[i]);
		}
	}
	return res;
}
/*
__global__ void initUG(int *x, int *e, int *prod) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int puiss = 2 * e[id];
	//int nbBlock;
	//int nbThread;
	int a = ceil(puiss/2);
	int b = floor(puiss/2)+1;
	int *res = (int*) malloc(a*sizeof(int));
	prodPuissG<<<1,a>>>(x, puiss, res, id);
	while (a > 1) {
		//nbBlock = ceil(puiss/1024);
		//nbThread = puiss%1024;
		b = floor(a/2)+1;
		a = ceil(a/2);
		prodPuissG<<<1,a>>>(res, b, res, id);
	}
	prod[id] = res[0];
}

int initU(int *x, int m, int *e) {
	int u;
	int *prodC;
	hipMalloc(&prodC, m*sizeof(int));
	int *somC;
	hipMalloc(&somC, (ceil(m/2))*sizeof(int));
	//int nbBlock = ceil(m/1024);
	//int nbThread = m%1024;
	int taille = m;
	int t;
	do {
		t = floor(taille/2)+1;
		taille = ceil(taille/2);
		sommeG<<<1,taille>>>(e, somC, t);
	} while (taille > 1);
	int somme;
	somme = 2 * somC[0];
	//hipMemcpy(&somme, somC[0], sizeof(int), hipMemcpyDeviceToHost);
	initUG<<<1,somme>>>(x, e, prodC);
	int *prod = (int*) malloc(m*sizeof(int));
	return u;
}*/
