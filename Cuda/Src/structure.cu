/*
 * structure.c
 *
 *  Created on: 6 févr. 2014
 *      Author: tony
 */
#include "header/structure.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * Alloue de l'espace memoire pour l'ensemble. Retourne un ensemble vide
 * et initialise size à 0.
 */
__host__ __device__ ensemble initEns(int *size){
	ensemble tp  = (ensemble) malloc(sizeof(struct cell));
	*size = 0;

	return tp;
}


/**
 * Ajoute (en tete) un couple dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL. !!ATTENTION POSSIBLE
 * FUITE MEMOIRE!!
 */
__host__ __device__ int addCouple(ensemble *ens, uint64_t x, uint64_t y,int *size){
	*size = *size+1;

	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));
	if(*size > 1){

		if(tp == NULL){
			printf("malloc nok size:= %i\n",*size);
			return -1;
		}
		if( memcpy(&tp,ens,(*size-1)*sizeof(ens)) == NULL){
			printf("erreur de recopie d'ensemble\n");
			return -1;
		}

	}

	tp[*size-1].ind.couple.x =x;
	tp[*size-1].ind.couple.y = y;

	*ens =tp;



	return 1;
}
/**
 * Ajoute (en tete) une valeur dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL
 */
__host__ __device__ int addVal(ensemble *ens, uint64_t x,int *size){
	/*	*size = (*size)+1;

	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));
	if(tp == NULL){
		return -1;
	}
	if(*size > 1){
	tp = ens;
	}
	tp[(*size) - 1].ind.val=x;
	printf("%i :: %i\n",tp[*size-1].ind.val,tp[*size-2].ind.val);

	ens = tp;


	return 1;
	 */
	*size = *size+1;

	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));
	if(*size > 1){

		tp=*ens;
		/*if( memcpy(&tp,ens,(*size-1)*sizeof(ens)) == NULL){
				printf("erreur de recopie d'ensemble\n");
				return -1;
			}*/

	}

	tp[*size-1].ind.val =x;


	*ens =tp;



	return 1;
}
