/*
 * structure.c
 */
#include "header/structure.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * Alloue de l'espace memoire pour l'ensemble. Retourne un ensemble vide
 * et initialise size à 0.
 */
__host__  ensemble initEns(int *size){
	ensemble tp  = (ensemble) malloc(sizeof(struct cell));
	*size = 0;
	return tp;
}


/**
 * Ajoute (en tete) un couple dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL. 
 */
__host__  int addCouple(ensemble *ens, uint64_t x, uint64_t y,int *size){
	*size = *size+1;

	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));
	if (*size > 1) {

		if (tp == NULL) {
			return -1;
		}
		if( memcpy(&tp,ens, (*size - 1) * sizeof(ens)) == NULL) {
			printf("erreur de recopie d'ensemble\n");
			return -1;
		}

	}

	tp[*size-1].ind.couple.x =x;
	tp[*size-1].ind.couple.y = y;

	*ens =tp;

	return 1;
}

/**
 * Ajoute (en tete) une valeur dans l'ensemble ens et retourne la taille de
 * l'ensemble si tout c'est bien passé sinon retourne NULL
 */
__host__  int addVal(ensemble *ens, uint64_t x,int *size){
	*size = *size+1;

	ensemble tp = (ensemble) malloc((*size)*sizeof(struct cell));

	if (*size > 1) {
		tp=*ens;
	}

	tp[*size-1].ind.val =x;
	*ens =tp;
	return 1;
}
