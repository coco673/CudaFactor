#include "hip/hip_runtime.h"
/*
 * TestFill.cu
 *
 *  Created on: 3 avr. 2014
 *      Author: groupeDev
 */

#include "fillEns.h"
#include "fillEnsemble.h"
#include "prime.h"
#include <unistd.h>
int TestfillEnsG(){
	int k = 0;
	int nbr = 201;
	int borne = ceil(exp(sqrt(2 * log(nbr) * log(log(nbr)))));

	int sqrtNRB = (int)sqrt(nbr);
	int *rand = (int *) malloc(k*sizeof(int));
	int *dev_rand;
	int *p = generatePrimeList(borne,&k);
	printf("k = %i\n",k);
	printf("sqrt = %i\n",sqrtNRB);
	int *dev_p;
	hiprandState_t *state = (hiprandState_t *)malloc(k*sizeof(hiprandState_t));
	hiprandState_t *dev_state ;
	int sizeR= 0;
	int *dev_sizeR;
	int sizeDiv = 0;
	int *dev_sizeDiv;
	int *matrix =(int *)malloc((k*k)*sizeof(int));
	int *dev_matrix;
	ensemble R;
	ensemble dev_R;
	ensemble Div ;
	ensemble dev_Div;


	R = (ensemble)malloc(k*sizeof(struct cell));
	Div = (ensemble) malloc(sizeof(struct cell));
	hipMalloc(&dev_R,k*sizeof(struct cell));
	hipMalloc(&dev_Div,sizeof(struct cell));
	hipMalloc(&dev_p,k*sizeof(int));
	hipMalloc(&dev_rand,k*sizeof(int));
	hipMalloc(&dev_sizeR,sizeof(int));
	hipMalloc(&dev_matrix,(k*k)*sizeof(int));

	hipMalloc(&dev_state,k*sizeof(hiprandState_t));
	hipMalloc(&dev_sizeDiv,sizeof(int));

	hipMemcpy(dev_sizeR,&sizeR,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,k*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_state,state,k*sizeof(hiprandState_t),hipMemcpyHostToDevice);
	hipMemcpy(dev_R,R,k*sizeof(struct cell),hipMemcpyHostToDevice);
	hipMemcpy(dev_Div,Div,sizeof(struct cell),hipMemcpyHostToDevice);
	hipMemcpy(dev_sizeDiv,&sizeDiv,sizeof(int),hipMemcpyHostToDevice);


	Generation<<<1,k>>>(dev_state,nbr, sqrtNRB,dev_rand);

	hipMemcpy(rand,dev_rand,k*sizeof(int),hipMemcpyDeviceToHost);
	for(int i = 0;i<k;i++){
		printf("blahaza %i\n",rand[i]);
	}
	printf("size Div = %i\n",sizeDiv);
	fillEnsR<<<1,k>>>(dev_state,dev_R,dev_sizeR,dev_Div,dev_sizeDiv,dev_p,k,dev_rand,nbr,dev_matrix);

	hipMemcpy(&sizeR,dev_sizeR,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(R,dev_R,k*sizeof(struct cell),hipMemcpyDeviceToHost);
	hipMemcpy(matrix,dev_matrix,(k*k)*sizeof(int),hipMemcpyDeviceToHost);


	for(int i = 0;i<k;i++){
		printf("x a la fin = %i y a la fin = %i\n",R[i].ind.couple.x,R[i].ind.couple.y);
	}
	for(int i = 0;i<k*k;i++){
		printf("mat = %i\n",matrix[i]);
	}
	/*hipFree(dev_R);
	hipFree(dev_Div);
	hipFree(dev_p);
	hipFree(dev_rand);
	hipFree(dev_sizeR);
	hipFree(dev_matrix);
	free(R);
	free(Div);
	free(rand);
	free(p);
	free(matrix);*/
}
