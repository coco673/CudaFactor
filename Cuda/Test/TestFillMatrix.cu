#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../src/header/fillMatrix.h"
#include "../src/header/prime.h"
#include "TestFillMatrix.h"

int main(int argc, char **argv) {
	int *yList = (int *) malloc(3 * sizeof(int));
	yList[0] = 3;
	yList[1] = 5;
	yList[2] = 6;
	int sizeYList = 3;
	int sizePrimeList;
	int *primeList = generatePrimeList(10, &sizePrimeList);
    for (int i = 0; i < sizePrimeList; i++) {
        printf("%d\n", primeList[i]);
    }
	/*int nbVect = pow(2, sizePrimeList);
	//taille des vecteurs
	int sizeVect = floor((double) log(sizePrimeList)/ (double) log(2)) + 1;
	int res = (int *) malloc(sizeYList * sizeVect * sizeof(int));
	int dev_res, dev_yList, dev_primeList;
	hipMalloc(dev_res, sizeYList * sizeVect * sizeof(int));
	hipMalloc(dev_yList, 3 * sizeof(int));
	hipMalloc(sizePrimeList * sizeof(int));
	hipMemcpy(dev_yList, yList, dev_yList, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_primeList, primeList, sizePrimeList * sizeof(int), hipMemcpyHostToDevice);
	fillMatrix<<<sizeYList, nbVect>>>(dev_yList, dev_primeList, sizeVect, dev_res);
	hipMemcpy(res, dev_res, sizeYList * sizeVect * sizeof(int), hipMemcpyDeviceToHost);
	assert(res[0] == 0);
	assert(res[1] == 1);
	for(int i = 2; i < sizeVect; i++) {
		assert(res[i] == 0);
	}
	assert(res[sizeVect] == 0);
	assert(res[sizeVect + 1] == 0);
	assert(res[sizeVect + 2] == 1);
	for (int i = sizeVect + 3; i < 2 * sizeVect; i++) {
		assert(res[i] == 0);
	}
	assert(res[2 * sizeVect] == 1);
	assert(res[2 * sizeVect + 1] == 1);
	for (int i = 2 * sizeVect + 2; i < 3 * sizeVect; i++) {
		assert(res[i] == 0);
	}*/
    int **result = fillMatrixNaif(yList, sizeYList, primeList, sizePrimeList);
    for (int i = 0; i < sizeYList; i++) {
        printf("yList[%d] : [", i);
        for (int j = 0; j < sizePrimeList; j++) {
            printf("%d", result[i][j]);
            if (j != sizePrimeList - 1) {
                printf(" : ");
            }
        }
        printf("]\n");
    }
    return EXIT_SUCCESS;
}
