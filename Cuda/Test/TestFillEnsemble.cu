#include "hip/hip_runtime.h"
/*
 *  TestFillEnsemble.c
 *  
 *
 *  Created by Tony on 21/02/2014.
 *  Copyright 2014 __MyCompanyName__. All rights reserved.
 *
 */

#include "TestFillEnsemble.h"
#include "../Src/header/prime.h"
#include "../Src/header/fillEnsemble.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int TestIsBSmooth(){
	int size;
	int borne = 100;
	int val = 200;
	int *list = generatePrimeList(borne,&size);
	size--;
	assert(isBSmooth(list, size, val));

	val = 10;
	assert(!isBSmooth(list, size, val));
	free(list);
	return 0;
}
int TestIsInEnsemble(){
	int size,i;
	ensemble e = initEns(&size);

	for (i = 0; i < 32; i++){
		addVal(&e,i,&size);
	}

	assert(isInEnsemble(e,12,size) == 1);
	assert(isInEnsemble(e,44,size) == 0);
	free(e);
	return 0;
}
__global__ void isInfKernel(int *dev_list,bool *result,int size,int val){
	int i =threadIdx.x;
	volatile __shared__ bool found;
	if(threadIdx.x == 0) found = false;
	__syncthreads();
	//Attention size prend une valeur de trop et bien superieur à la borne !!
	//TODO Rectifier la fonction generatePrimeList
	if(found== false  && i < size-1){
		int inf = isInf(dev_list,size-1,val);
		if(inf){
			found = true;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

int TestIsInf(){
	int borne = 20;
	int val = 8;
	int size;

	int *list = generatePrimeList(borne,&size);
	int *dev_list;
	bool *dev_result;
	bool *result=(bool *) malloc(sizeof(bool));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);


	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == true);

	free(result);
	hipFree(dev_result);

	val = 200;
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == false);


	hipFree(dev_result);
	hipFree(dev_list);

	free(list);
	return 0;
}
__global__ void IsBSmoothKernel(int *list,int size, int y,int *result){

	isBSmoothG(list,size,y,result);
}
int TestIsBSmoothG(){
	int borne = 99;
	int val = 20;
	int size;
	int *list = generatePrimeList(val,&size);
	//Rectification temporaire
	size--;
	int *dev_list;
	int *dev_result;
	int *result=(int *) malloc(sizeof(int));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 1);

	hipFree(dev_list);
	hipFree(dev_result);
	free(result);
	free(list);
	size = 0;
	val = 200;
	list = generatePrimeList(val,&size);
	//Rectification temporaire
	size--;
	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	result = (int *) malloc(sizeof(int));
	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 0);

	hipFree(dev_list);
	hipFree(dev_result);
	free(result);
	free(list);
	return 0;
}

__global__ void IsInEnsembleKernel(ensemble ens,int size, int y,int *result){

	isInEnsembleG(ens,y,size,result);
}

int TestIsInEnsembleG(){

	int size;
	ensemble ens = initEns(&size);
	ensemble dev_ens;
	int val = 12;

	int *dev_result;
	int i;
	for (i = 0; i < 16; i++){
		addVal(&ens,i,&size);

	}

	int *result=(int *) malloc(sizeof(int));
	hipMalloc(&dev_ens,size*sizeof(struct cell));
	hipMalloc(&dev_result,sizeof(int));

	hipMemcpy(dev_ens,ens,size*sizeof(struct cell),hipMemcpyHostToDevice);
	IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 1);

	hipFree(dev_result);
	val = 20045;
	result=(int *) malloc(size*sizeof(int));

	hipMalloc(&dev_result,size*sizeof(int));
	IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
	hipMemcpy(result,dev_result,size*sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 0);

	hipFree(dev_result);
	free(ens);
	hipFree(dev_ens);

	return 0;
}


int TestfillEnsemble(){
	int size;
	int nbr = 257;
	int borne = 10;
	ensemble div = initEns(&size);
	ensemble e ;
	fillEnsemble(e,nbr,borne,div,size);

	return 0;
}
int TestfillEnsembleG(){
	int sizediv;
	int *size=(int *)malloc(sizeof(int));

	int *dev_size;
	int k;
	int nbr = 257349;
	int borne = 10;
	ensemble div = initEns(&sizediv);
	ensemble dev_div;
	ensemble r ;
	ensemble dev_r;
	int *p =generatePrimeList(borne,&k);
	k--;
	int *dev_p;

	hipMalloc(&dev_p,k*sizeof(int));
	hipMalloc(&dev_size,sizeof(int));
	hipMalloc(&dev_r,sizeof(struct cell));
	hipMalloc(&dev_div,sizediv*sizeof(struct cell));
	hipMalloc(&dev_div,sizediv*sizeof(struct cell));

	hipMemcpy(dev_p,p,k*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,k*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_div,div,sizediv*sizeof(struct cell),hipMemcpyHostToDevice);
	hipMemcpy(dev_size,size,sizeof(int),hipMemcpyHostToDevice);

	fillEnsembleG<<<1,k>>>(dev_r,dev_p,k,nbr,borne,dev_div,sizediv,dev_size);

	hipMemcpy(size,dev_size,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(r,dev_r,(*size)*sizeof(struct cell),hipMemcpyDeviceToHost);
printf("size %i\n",*size);
for (int i =0; i<(*size);i++){
	printf("x = %i ; y= %i\n",r[i].ind.couple.x,r[i].ind.couple.y);

}
	return 0;
}

__global__ void setup_kernelGen(int *rand){
	int id = threadIdx.x;
	int nbr = 29;
	int racN = 12;
	int *tprand = (int*) malloc(gridDim.x*sizeof(int));

	hiprandState_t *local = (hiprandState_t*)malloc((blockDim.x*gridDim.x)*sizeof(hiprandState_t));
	setup_kernel(local);
	generate(local,tprand,nbr,racN);
	rand[id] = tprand[id];

}
int TestGenerateOnce(){
	int *rand = (int *)malloc(10*sizeof(int));
	int *dev_rand;

	hipMalloc(&dev_rand,10*sizeof(int));


	setup_kernelGen<<<1,10>>>(dev_rand);
	hipMemcpy(rand,dev_rand,10*sizeof(int),hipMemcpyDeviceToHost);

}
