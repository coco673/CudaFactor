#include "hip/hip_runtime.h"
/*
 *  TestFillEnsemble.c
 *  
 *
 *  Created by Tony on 21/02/2014.
 *  Copyright 2014 __MyCompanyName__. All rights reserved.
 *
 */

#include "TestFillEnsemble.h"
#include "../Src/header/prime.h"
#include "../Src/header/fillEnsemble.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int TestIsBSmooth(){
	int size;
	int borne = 100;
	int val = 200;
	int *list = generatePrimeList(borne,&size);
	size--;
	assert(isBSmooth(list, size, val));

	val = 10;
	assert(!isBSmooth(list, size, val));
	free(list);
	return 0;
}
int TestIsInEnsemble(){
	int size,i;
	ensemble e = initEns(&size);

	for (i = 0; i < 32; i++){
		addVal(e,i,&size);
	}
	assert(isInEnsemble(e,12,size) == 1);
	assert(isInEnsemble(e,44,size) == 0);
	free(e);
	return 0;
}
__global__ void isInfKernel(int *dev_list,bool *result,int size,int val){
	int i =threadIdx.x;
	volatile __shared__ bool found;
	if(threadIdx.x == 0) found = false;
	__syncthreads();
	//Attention size prend une valeur de trop et bien superieur à la borne !!
	//TODO Rectifier la fonction generatePrimeList
	if(found== false  && i < size-1){
		int inf = isInf(dev_list,size-1,val);
		if(inf){
			found = true;
			*result = found;
		}
		__syncthreads();
		*result = found;
	}
}

int TestIsInf(){
	int borne = 20;
	int val = 8;
	int size;

	int *list = generatePrimeList(borne,&size);
	int *dev_list;
	bool *dev_result;
	bool *result=(bool *) malloc(sizeof(bool));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);


	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == true);

	free(result);
	hipFree(dev_result);

	val = 200;
	hipMalloc(&dev_result,sizeof(bool));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	isInfKernel<<<1,size>>>(dev_list,dev_result,size,val);
	hipMemcpy(result,dev_result,sizeof(bool),hipMemcpyDeviceToHost);
	assert(*result == false);


	hipFree(dev_result);
	hipFree(dev_list);
	//free(result);
	free(list);
	return 0;
}
__global__ void IsBSmoothKernel(int *list,int size, int y,int *result){

	isBSmoothG(list,size,y,result);
}
int TestIsBSmoothG(){
	int borne = 99;
	int val = 20;
	int size;
	int *list = generatePrimeList(val,&size);
	//Rectification temporaire
	size--;
	int *dev_list;
	int *dev_result;
	int *result=(int *) malloc(sizeof(int));

	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 1);

	hipFree(dev_list);
	hipFree(dev_result);
	free(result);
	free(list);
	size = 0;
	val = 200;
	list = generatePrimeList(val,&size);
	//Rectification temporaire
	size--;
	hipMalloc(&dev_list,size*sizeof(int));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_list,list,size*sizeof(int),hipMemcpyHostToDevice);

	result = (int *) malloc(sizeof(int));
	IsBSmoothKernel<<<1,size>>>(dev_list,size,borne,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);

	assert(*result == 0);

	hipFree(dev_list);
	hipFree(dev_result);
	free(result);
	free(list);
	return 0;
}

__global__ void IsInEnsembleKernel(ensemble ens,int size, int y,int *result){

	isInEnsembleG(ens,y,size,result);

}

int TestIsInEnsembleG(){

	int size;
	ensemble ens = initEns(&size);
	ensemble dev_ens;
	int val = 22;

	int *dev_result;
	int i;
//TODO revoir addVall
	for (i = 0; i < 16; i++){
		addVal(ens,i,&size);
		printf("sizeof %i ::ens[%i] = %i :: size = %i :: i = %i\n",sizeof(ens),i,ens[i].ind.val,size,i);

	}
	int *result=(int *) malloc(sizeof(int));
	hipMalloc(&dev_ens,size*sizeof(struct cell));
	hipMalloc(&dev_result,sizeof(int));
	hipMemcpy(dev_ens,ens,size*sizeof(struct cell),hipMemcpyHostToDevice);
	IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
	hipMemcpy(result,dev_result,sizeof(int),hipMemcpyDeviceToHost);
	printf("%i\n",*result);
	assert(*result == 1);


	//free(result);
	hipFree(dev_result);
	val = 20045;
	result=(int *) malloc(size*sizeof(int));

	hipMalloc(&dev_result,size*sizeof(int));
	printf("val : %i\n",val);
	IsInEnsembleKernel<<<1,size>>>(dev_ens,size,val,dev_result);
	hipMemcpy(result,dev_result,size*sizeof(int),hipMemcpyDeviceToHost);
	printf("%i\n",*result);
	assert(*result == 0);


	hipFree(dev_result);
	free(ens);
	hipFree(dev_ens);

	return 0;
}


int TestfillEnsemble(){
	int size;
	int nbr = 257;
	int borne = 10;
	ensemble div = initEns(&size);
	ensemble e ;
	fillEnsemble(e,nbr,borne,div,size);

	return 0;
}
int TestfillEnsembleG(){
	int sizediv;
	int *size=(int *)malloc(sizeof(int));

	int *dev_size;
	int k;
	int nbr = 257349;
	int borne = 10;
	ensemble div = initEns(&sizediv);
	ensemble r =initEns(size);
	ensemble dev_r;
	int *p =generatePrimeList(borne,&k);
	k--;


	hipMalloc(&dev_size,sizeof(int));
	hipMalloc(&dev_r,sizeof(struct cell));

	hipMemcpy(dev_r,r,sizeof(struct cell),hipMemcpyHostToDevice);
	hipMemcpy(dev_size,size,sizeof(int),hipMemcpyHostToDevice);

	fillEnsembleG<<<5,1>>>(dev_r,p,k,nbr,borne,div,sizediv,dev_size);

	hipMemcpy(size,dev_size,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(r,dev_r,*size*sizeof(struct cell),hipMemcpyDeviceToHost);

	return 0;
}

__global__ void setup_kernelGen(int *rand){
	int id = threadIdx.x;
	int nbr = 29;
	int racN = 12;
	int *tprand = (int*) malloc(gridDim.x*sizeof(int));
	hiprandState_t *local = (hiprandState_t*)malloc((blockDim.x*gridDim.x)*sizeof(hiprandState_t));
	setup_kernel(local);
	generate(local,tprand,nbr,racN);
	rand[id] = tprand[id];
	for (int i= 0;i<10;i++){
			printf("%i\n",rand[i]);

		}
}
int TestGenerateOnce(){
	int *rand = (int *)malloc(10*sizeof(int));
	int *dev_rand;

	hipMalloc(&dev_rand,10*sizeof(int));


	setup_kernelGen<<<1,10>>>(dev_rand);
	hipMemcpy(rand,dev_rand,10*sizeof(int),hipMemcpyDeviceToHost);

	for (int i= 0;i<10;i++){
		printf("%i\n",rand[i]);

	}
}
